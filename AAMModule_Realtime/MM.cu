#include "hip/hip_runtime.h"
#ifndef GPUBASIC
#define GPUBASIC


#include <crtdbg.h>
#include <hipblas.h>
#include <cula.h>
#include <fstream>
#include <iostream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cstdio>



using namespace std;
using namespace thrust;


#ifdef _DEBUG
#define CUDA_CALL( call ) do { \
	hipError_t err; \
	err = (call); \
	if( err != hipSuccess ) \
	{ \
		fprintf( stderr, "error in CUDA call in file '%s', line: %d\n" \
					"error %d: %s\n", \
							__FILE__, __LINE__, \
					err, hipGetErrorString( err ) ); \
		if(::_CrtDbgReport(_CRT_ASSERT, __FILE__, __LINE__, NULL, NULL)==1) \
		{ \
			::_CrtDbgBreak(); \
		} \
	} \
} while(0)

#define CUBLAS_CALL( call ) do { \
	hipblasStatus_t err; \
	err = (call); \
	if( err != HIPBLAS_STATUS_SUCCESS ) \
	{ \
	fprintf( stderr, "error in CUBLAS call in file '%s', line: %d\n" \
	"error %d:\n", \
	__FILE__, __LINE__, \
	err); \
	if(::_CrtDbgReport(_CRT_ASSERT, __FILE__, __LINE__, NULL, NULL)==1) \
		{ \
		::_CrtDbgBreak(); \
		} \
	} \
} while(0)

#define CULA_CALL( call ) do { \
	culaStatus err; \
	culaInfo info; \
	char err_string[1000]; \
	err = (call); \
	if( err != culaNoError ) \
	{ \
	info = culaGetErrorInfo(); \
	culaGetErrorInfoString(err, info, err_string, 1000); \
	fprintf( stderr, "error in CULA call in file '%s', line: %d\n" \
	"error %d: %s, %s\n", \
	__FILE__, __LINE__, \
	err, culaGetStatusString( err ), err_string ); \
	if(::_CrtDbgReport(_CRT_ASSERT, __FILE__, __LINE__, NULL, NULL)==1) \
		{ \
		::_CrtDbgBreak(); \
		} \
	} \
} while(0)

#define CUDA_ASSERT( call ) do { \
	if(!(call))  \
	{ \
	printf("assertion in CUDA call in file '%s', line: %d\n" \
	__FILE__, __LINE__); \
	return; }\
} while(0)

#else		// FOR RELEASE
/*
 * CUDA error handling macro
 */
#define CUDA_CALL( call ) \
        { \
            hipError_t err; \
            err = (call); \
            if( err != hipSuccess ) \
            { \
                fprintf( stderr, "error in CUDA call in file '%s', line: %d\n" \
                        "%s\nerror %d: %s\nterminating!\n", \
                        __FILE__, __LINE__, #call, \
                        err, hipGetErrorString( err ) ); \
                exit( ~0 ); \
            } \
        }

#define CUBLAS_CALL( call ) \
		{ \
		hipblasStatus_t err; \
		err = (call); \
		if( err != HIPBLAS_STATUS_SUCCESS ) \
			{ \
			fprintf( stderr, "error in CUBLAS call in file '%s', line: %d\n" \
			"%s\nerror %d: %s\nterminating!\n", \
			__FILE__, __LINE__, #call, \
			err ); \
			exit( ~0 ); \
			} \
		}

#define CULA_CALL( call ) \
		{ \
		culaStatus err; \
		culaInfo info; \
		char err_string[1000]; \
		char *err_type; \
		err = (call); \
		if( err != culaNoError ) \
			{ \
			info = culaGetErrorInfo(); \
			err_type = "CULA Error"; \
			if(err == culaArgumentError) \
			{ \
				err_type = "Argument error"; \
			} \
			else \
			if(err == culaDataError) \
			{ \
				err_type = "Data error"; \
			} \
			culaGetErrorInfoString(err, info, err_string, 1000); \
			fprintf( stderr, "%s in CULA call in file '%s', line: %d\n" \
			"%s\nerror %d: %s, %s\nterminating!\n", \
			err_type, __FILE__, __LINE__, #call, \
			err, culaGetStatusString( err ), err_string); \
			exit( ~0 ); \
			} \
		}

#define CUDA_ASSERT( call )  

#define CUDA_RASSERT( call )  do { \
	if(!(call))  \
	{ \
	printf("assertion in CUDA call\n"); \
	return; }\
} while(0)

#endif

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__);            \
	return EXIT_FAILURE;}} while(0)

// let *destDevice = src, where *destDevice locates in GPU
template<typename T>
void copyVarToDevice(T *destDevice, const T& src);
// return *valInDevice in host
template<typename T>
T getVarFromDevice(T *valInDevice);

// let *destDevice = src, where *destDevice locates in GPU
template<typename T>
void copyVarToDevice(T *destDevice, const T& src)
{
	CUDA_CALL(hipMemcpy(destDevice, &src, sizeof(T), hipMemcpyHostToDevice));
}

// return *valInDevice in host
template<typename T>
T getVarFromDevice(T *valInDevice)
{
	T val;
	CUDA_CALL(hipMemcpy(&val, valInDevice, sizeof(T), hipMemcpyDeviceToHost));
	return val;
}

template<typename T>
const T getVarFromDevice(const T *valInDevice)
{
	T val;
	CUDA_CALL(hipMemcpy(&val, valInDevice, sizeof(T), hipMemcpyDeviceToHost));
	return val;
}

template<typename T>
void NewArrayInDevice(T **ptr, int numElement)
{
	CUDA_CALL(hipMalloc(ptr, sizeof(T) * numElement));
}

template<typename T>
T* NewArrayInDevice(int numElement, const T* srcArray = NULL)
{
	T* ret;
	CUDA_CALL(hipMalloc(&ret, sizeof(T) * numElement));
	if(srcArray)
	{
		CUDA_CALL( hipMemcpy(ret, srcArray, sizeof(T)*numElement, hipMemcpyHostToDevice) );
	}
	return ret;
}

template<typename T>
void deleteArrayInDevice(T *ptr)
{
	CUDA_CALL(hipFree(ptr));
}

template<typename T>
void copyArrayToHost(T* dstArray, const T* srcArrayGPU, int copyElem)
{
	CUDA_CALL(hipMemcpy(dstArray, srcArrayGPU, sizeof(T)*copyElem, hipMemcpyDeviceToHost));
}

template<typename T>
void copyArrayToDevice(T* dstArray, const T* srcArrayGPU, int copyELem)
{
	CUDA_CALL(hipMemcpy(dstArray, srcArrayGPU, sizeof(T)*copyELem, hipMemcpyHostToDevice));
}

template<typename T>
T* getArrayFromDevice(const T* srcArrayGPU, int copyELem)
{
	T* ret = new T[copyELem];
	CUDA_CALL(hipMemcpy(ret, srcArrayGPU, sizeof(T)*copyELem, hipMemcpyHostToDevice));
	return ret;
}


double* DeviceDoubleArray(int size);
double* DeviceDoubleArray(int size, const double* host_array);
double* DeviceDoubleArray(int size, double value);
double* HostDoubleArray(int size, const double* device_array);
int* DeviceIntArray(int size, const int* host_array);
int* DeviceIntArray(int size, int value);
int* HostIntArray(int size, const int* device_array);

__host__ __device__ inline bool operator==(const float4& a, const float4& b)
{
	if(a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w)
	{
		return true;
	}
	return false;
}

__host__ __device__ inline bool operator!=(const float4& a, const float4& b)
{
	if(a.x != b.x || a.y != b.y || a.z != b.z || a.w != b.w)
	{
		return true;
	}
	return false;
}

__host__ __device__ inline void printData(const char *name, const float4& f, const char *tail)
{
	printf("%s (%0.1f %0.1f %0.1f %0.1f) %s", name, f.x, f.y, f.z, f.w, tail);
}

__host__ __device__ inline float Dis2(const float4& p1, const float4& p2) 
{ 
	float px = p1.x - p2.x;
	float py = p1.y - p2.y;
	float pz = p1.z - p2.z;
	return px*px + py*py + pz*pz; 
}

__host__ __device__ inline float Dis(const float4& p1, const float4& p2) 
{ 
	float px = p1.x - p2.x;
	float py = p1.y - p2.y;
	float pz = p1.z - p2.z;
	return sqrt(px*px + py*py + pz*pz); 
}

__device__ inline float my_round(float number)
{
	return number < 0.0 ? ceil(number - 0.5) : floor(number + 0.5);
}

__device__ inline int dataClip(int data, int minVal, int maxVal)
{
	return ( (data < minVal) ? minVal : (data > maxVal) ? maxVal : data );
}

//__host__ __device__ inline float4 operator+(const float4& a, const float4& b)
//{
//	return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
//}

//__host__ __device__ inline float4 operator/(const float4& a, const float& b)
//{
//	return make_float4(a.x/b, a.y/b, a.z/b, a.w/b);
//}
inline std::ostream& operator<<(std::ostream& os, const float4& p) 
{ 
	os<<p.x<<" "<<p.y<<" "<<p.z<<" "<<p.w; 
	return os; 
}

template<typename Type>
void MemsetCuda(Type* ptr, Type value, int elementCount)
{
	CUDA_CALL( hipMemset(ptr, value, sizeof(Type)*elementCount) );
}

inline void showCUDAMemoryUsage(const char* str = NULL)
{
	size_t free_byte ;
	size_t total_byte ;
	hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
	if ( hipSuccess != cuda_status ){
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
		exit(1);
	}

	double free_db = (double)free_byte ;
	double total_db = (double)total_byte ;
	double used_db = total_db - free_db ;
	if(str != NULL)
	{
		printf("%s GPU memory usage: \n\tused = %f \n\tfree = %f MB \n\ttotal = %f MB\n", str,
			used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	} else
	{
		printf("GPU memory usage: \n\tused = %f \n\tfree = %f MB \n\ttotal = %f MB\n",
			used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	}
}
hipblasHandle_t blas_handle_;
int MAX_COUNT = 50;
float *raw_gpu;
device_vector<float> data_gpu;
extern "C" void aa(int rows,int cols,int row_pitch,int numF,vector<float> &data_cpu)
{
	
	
	
	
		
	/*for(int i=0;i<rows;i++)
	{
		for(int j=0;j<cols;j++)
		{
			cout<<data_cpu[i*cols+j]<<" ";		
		}	
		cout<<endl;
	}*/
	
//	device_vector<float> data_gpu_trans(MAX_COUNT);
//	float *raw_gpu_trans = raw_pointer_cast(&data_gpu_trans[0]);

	// fill data_cpu
	CUDA_CALL(hipMemcpy(raw_gpu, &data_cpu[0], MAX_COUNT*sizeof(float), hipMemcpyHostToDevice ));

	// data_gpu: row_pitch * numF, rows*cols
	//CULA_CALL( culaDeviceSgeTranspose(numF, row_pitch, 
	//	raw_gpu, numF,
	//	raw_gpu_trans, row_pitch) );
		


	int m = cols;
	int n = cols;
	int k = rows;
	int lda = numF;
	int ldb = numF;
	int ldc = numF;
	
	float alpha = 1.0f;
	float beta = 0;

	
	// tmpC = dataA' * dataA
	CUBLAS_CALL( hipblasSgemm(blas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
		m, n, k, 
		&alpha, raw_gpu, lda, 
		raw_gpu, ldb,
		&beta,
		raw_gpu, ldc) );

	CUDA_CALL(hipMemcpy(&data_cpu[0], raw_gpu, MAX_COUNT*sizeof(float), hipMemcpyDeviceToHost ));
	

	
/*	for(int i=0;i<cols;i++)
	{
		for(int j=0;j<cols;j++)
		{
			cout<<data_cpu[i*cols+j]<<" ";		
		}	
		cout<<endl;
	}*/

}

extern "C" void init()
{
	
	CUBLAS_CALL(hipblasCreate(&blas_handle_));
	CULA_CALL(culaInitialize());
//	data_gpu.resize(MAX_COUNT);
	//raw_gpu = raw_pointer_cast(&data_gpu[0]);
}

extern "C"  void endSection()
{
	CUBLAS_CALL( hipblasDestroy(blas_handle_) );
	culaShutdown();
	
}

#endif