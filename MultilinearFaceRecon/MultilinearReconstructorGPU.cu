#include "hip/hip_runtime.h"
#include "MultilinearReconstructorGPU.cuh"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include "Kinect/KinectUtils.h"
#include "Utils/Timer.h"
#include "Utils/stringutils.h"
#include "Utils/utility.hpp"

#include "Elements_GPU.h"
#include "utils_GPU.cuh"
#include "numerical_algorithms.cuh"


#define FBO_DEBUG_GPU 0
#define KERNEL_DEBUG 0
#define OUTPUT_ICPC 0
#define FAST_RENDER 1

MultilinearReconstructorGPU::MultilinearReconstructorGPU():
	d_tu0(nullptr), d_tu1(nullptr), d_tm0(nullptr), d_tm1(nullptr),
	d_tplt(nullptr), d_mesh(nullptr), d_tm0RT(nullptr), d_tm1RT(nullptr),
	d_fptsIdx(nullptr), d_q2d(nullptr), d_q(nullptr), 
	d_colordata(nullptr), d_depthdata(nullptr),
	d_targetLocations(nullptr), d_RTparams(nullptr),
	d_A(nullptr), d_b(nullptr), d_meshtopo(nullptr), d_meshverts(nullptr)
{
	// set device
	cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
	checkCudaState();

	hipSetDeviceFlags(hipDeviceMapHost);

	PhGUtils::message("initializing CULA ...");
	culaInitialize();
	checkCudaState();
	PhGUtils::message("creating CUDA stream ...");
	hipStreamCreate(&mystream);
	checkCudaState();

	meanX = meanY = meanZ = 0;

	// initialize offscreen renderer
	initRenderer();

	// should be large enough
	NumericalAlgorithms::initialize(50, 16384);
	
	// initialize members
	init();

	initializeWeights();

	// process the loaded data
	preprocess();
	
	hipDeviceSynchronize();
}

MultilinearReconstructorGPU::~MultilinearReconstructorGPU() {
	// release resources
	hipDeviceReset();
}

__host__ void MultilinearReconstructorGPU::setPose(const float* params) {
	for(int i=0;i<7;i++) h_RTparams[i] = params[i];
	PhGUtils::printArray(params, 7);
	checkCudaErrors(hipMemcpy(d_RTparams, params, sizeof(float)*7, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_meanRT, params, sizeof(float)*7, hipMemcpyHostToDevice));
}

__host__ void MultilinearReconstructorGPU::setIdentityWeights(const Tensor1<float>& t) {
	// copy to GPU
	checkCudaErrors(hipMemcpy(d_Wid, t.rawptr(), sizeof(float)*ndims_wid, hipMemcpyHostToDevice));
	// update tensor tm0
	hipblasSgemv('N', ndims_wexp * ndims_pts, ndims_wid, 1.0, d_tu0, ndims_wexp * ndims_pts, d_Wid, 1, 0, d_tm0, 1);

	// update distance map
	hipblasSgemv('N', npts_mesh, ndims_wid, 1.0, d_rawdistmap, npts_mesh, d_Wid, 1, 0, d_distmap, 1);
	checkCudaState();
	writeback(d_distmap, npts_mesh, "d_distmap.txt");
	writeback(d_rawdistmap, ndims_wid, npts_mesh, "d_rawdistmap.txt");

	// and the template mesh
	hipblasSgemv('T', ndims_wid, ndims_pts, 1.0, d_tm1, ndims_wid, d_Wid, 1, 0.0, d_tplt, 1);
}

__host__ void MultilinearReconstructorGPU::setExpressionWeights(const Tensor1<float>& t) {
	t.print();
	// copy to GPU
	checkCudaErrors(hipMemcpy(d_Wexp, t.rawptr(), sizeof(float)*ndims_wexp, hipMemcpyHostToDevice));
	// update tensor tm1
	hipblasSgemv('N', ndims_wid * ndims_pts, ndims_wexp, 1.0, d_tu1, ndims_wid * ndims_pts, d_Wexp, 1, 0, d_tm1, 1);

	// and the template mesh
	hipblasSgemv('T', ndims_wid, ndims_pts, 1.0, d_tm1, ndims_wid, d_Wid, 1, 0.0, d_tplt, 1);
}

__host__ void MultilinearReconstructorGPU::preprocess() {
	PhGUtils::message("preprocessing the input data...");

	// process distance map
	hipblasSgemv('N', npts_mesh, ndims_wid, 1.0, d_rawdistmap, ndims_pts, d_mu_wid0, 1, 0, d_distmap, 1);

	// process the identity prior

	// invert sigma_wid
	int* ipiv;
	checkCudaErrors(hipMalloc((void**) &ipiv, sizeof(int)*ndims_wid));
	culaDeviceSgetrf(ndims_wid, ndims_wid, d_sigma_wid, ndims_wid, ipiv);
	culaDeviceSgetri(ndims_wid, d_sigma_wid, ndims_wid, ipiv);
	checkCudaErrors(hipFree(ipiv));

	// multiply inv_sigma_wid to mu_wid
	hipblasSgemv('N', ndims_wid, ndims_wid, 1.0, d_sigma_wid, ndims_wid, d_mu_wid, 1, 0.0, d_mu_wid_weighted, 1); 
	
	// scale inv_sigma_wid by w_prior_id
	hipblasSscal(ndims_wid*ndims_wid, w_prior_id, d_sigma_wid, 1);

	// scale mu_wid by w_prior_id
	hipblasSscal(ndims_wid, w_prior_id, d_mu_wid_weighted, 1);

	// copy back the inverted matrix to check correctness
	writeback(d_sigma_wid, ndims_wid*ndims_wid, "invswid.txt");

	// process the expression prior

	// invert sigma_wexp
	checkCudaErrors(hipMalloc((void**) &ipiv, sizeof(int)*ndims_wexp));
	culaDeviceSgetrf(ndims_wexp, ndims_wexp, d_sigma_wexp, ndims_wexp, ipiv);
	culaDeviceSgetri(ndims_wexp, d_sigma_wexp, ndims_wexp, ipiv);
	checkCudaErrors(hipFree(ipiv));

	// multiply inv_sigma_wexp to mu_wexp
	hipblasSgemv('N', ndims_wexp, ndims_wexp, 1.0, d_sigma_wexp, ndims_wexp, d_mu_wexp, 1, 0, d_mu_wexp_weighted, 1);

	// scale inv_sigma_wexp by w_prior_exp
	hipblasSscal(ndims_wexp*ndims_wexp, w_prior_exp, d_sigma_wexp, 1);

	// scale mu_wexp by w_prior_exp
	hipblasSscal(ndims_wexp, w_prior_exp, d_mu_wexp_weighted, 1); 

	writeback(d_sigma_wexp, ndims_wexp*ndims_wexp, "invswexp.txt");
	PhGUtils::message("done.");

	// initialize Wid and Wexp
	checkCudaErrors(hipMemcpy(d_Wid, d_mu_wid0, sizeof(float)*ndims_wid, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_Wexp, d_mu_wexp0, sizeof(float)*ndims_wexp, hipMemcpyDeviceToDevice));

	// initialize tm0, tm1

	// tm0 = tu0 * Wid, use cublas
	// tu0: ndims_wid * (ndims_wexp * ndims_pts) matrix, each row corresponds to an identity
	//		inside each row, the vertices are arranged by expression
	//		That is, a row in tu0 can be see as a row-major matrix where each row corresponds to an expression
	// tm0: a row-major matrix where each row corresponds to an expression
	hipblasSgemv('N', ndims_wexp * ndims_pts, ndims_wid, 1.0, d_tu0, ndims_wexp * ndims_pts, d_Wid, 1, 0, d_tm0, 1);
	writeback(d_tm0, ndims_wexp, ndims_pts, "tm0.txt");

	// tm1 = tu1 * Wexp, use cublas
	// tu1: ndims_wexp * (ndims_wid * ndims_pts) matrix, each row corresponds to an expression
	//		inside each row, the vertices are arraged using index-major
	//		That is, a row in tu1 can be see as a column-major matrix where each column corresponds to an identity
	// tm1: a column-major matrix where each column corresponds to an identity
	hipblasSgemv('N', ndims_wid * ndims_pts, ndims_wexp, 1.0, d_tu1, ndims_wid * ndims_pts, d_Wexp, 1, 0, d_tm1, 1);
	writeback(d_tm1, ndims_pts, ndims_wid, "tm1.txt");

	// create template mesh
	// tplt = tm1 * Wid, use cublas
	hipblasSgemv('T', ndims_wid, ndims_pts, 1.0, d_tm1, ndims_wid, d_Wid, 1, 0.0, d_tplt, 1);
	writeback(d_tplt, ndims_pts/3, 3, "tplt.txt");
}

__host__ void MultilinearReconstructorGPU::init() {
	showCUDAMemoryUsage();
	// read the core tensor
	PhGUtils::message("Loading core tensor ...");
	const string filename = "../Data/blendshape/core.bin";

	Tensor3<float> core;
	core.read(filename);
	core_dim[0] = core.dim(0), core_dim[1] = core.dim(1), core_dim[2] = core.dim(2);
	int totalSize = core_dim[0] * core_dim[1] * core_dim[2];

	tmesh.resize(core_dim[2]);
	npts_mesh = core_dim[2]/3;

	// unfold it
	Tensor2<float> tu0 = core.unfold(0), tu1 = core.unfold(1);

	PhGUtils::message("transferring the unfolded core tensor to GPU ...");

#if 1
	checkCudaErrors(hipHostAlloc((void**) &h_tu0, sizeof(float)*totalSize, hipHostMallocMapped));
	memcpy(h_tu0, tu0.rawptr(), sizeof(float)*totalSize);
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_tu0, h_tu0, 0));

	checkCudaErrors(hipHostAlloc((void**) &h_tu1, sizeof(float)*totalSize, hipHostMallocMapped));
	memcpy(h_tu1, tu1.rawptr(), sizeof(float)*totalSize);
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_tu1, h_tu1, 0));
#else
	// transfer the unfolded core tensor to GPU
	checkCudaErrors(hipMalloc((void **) &d_tu0, sizeof(float)*totalSize));
	checkCudaErrors(hipMemcpy(d_tu0, tu0.rawptr(), sizeof(float)*totalSize, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void **) &d_tu1, sizeof(float)*totalSize));
	checkCudaErrors(hipMemcpy(d_tu1, tu1.rawptr(), sizeof(float)*totalSize, hipMemcpyHostToDevice));
#endif

	PhGUtils::message("done.");
	showCUDAMemoryUsage();

	PhGUtils::message("allocating memory for computation (tensors) ...");
	// allocate memory for the tm0, tm1, tm0RT, tm1RT, tplt
	checkCudaErrors(hipMalloc((void **) &d_tm0, sizeof(float)*core_dim[1]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tm0RT, sizeof(float)*core_dim[1]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tm1, sizeof(float)*core_dim[0]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tm1RT, sizeof(float)*core_dim[0]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tplt, sizeof(float)*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_mesh, sizeof(float)*core_dim[2]));	
	checkCudaErrors(hipMemset(d_mesh, 0, sizeof(float)*core_dim[2]));

	Tensor2<float> distmap;
	distmap.read("../Data/blendshape/distmap.bin");
	checkCudaErrors(hipMalloc((void **) &d_rawdistmap, sizeof(float)*npts_mesh*core_dim[0]));
	checkCudaErrors(hipMemcpy(d_rawdistmap, distmap.rawptr(), sizeof(float)*npts_mesh*core_dim[0], hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void **) &d_distmap, sizeof(float)*npts_mesh));
	showCUDAMemoryUsage();
	writeback(d_rawdistmap, core_dim[0], npts_mesh, "d_rawdistmap.txt");

	// read the prior
	PhGUtils::message("Loading prior data ...");

	// identity prior
	PhGUtils::message("Loading identity prior data ...");
	const string fnwid  = "../Data/blendshape/wid.bin";

	ifstream fwid(fnwid, ios::in | ios::binary );
	fwid.read(reinterpret_cast<char*>(&ndims_wid), sizeof(int));
	cout << "identity prior dim = " << ndims_wid << endl;
	vector<float> mu_wid0, mu_wid, sigma_wid;
	mu_wid0.resize(ndims_wid);
	mu_wid.resize(ndims_wid);
	sigma_wid.resize(ndims_wid*ndims_wid);

	fwid.read(reinterpret_cast<char*>(&(mu_wid0[0])), sizeof(float)*ndims_wid);
	fwid.read(reinterpret_cast<char*>(&(mu_wid[0])), sizeof(float)*ndims_wid);
	fwid.read(reinterpret_cast<char*>(&(sigma_wid[0])), sizeof(float)*ndims_wid*ndims_wid);

	fwid.close();

	PhGUtils::message("identity prior loaded.");
	PhGUtils::message("transferring identity prior to GPU ...");

	// transfer the identity prior to GPU
	checkCudaErrors(hipMalloc((void **) &d_mu_wid0, sizeof(float)*ndims_wid));
	checkCudaErrors(hipMemcpy(d_mu_wid0, &(mu_wid0[0]), sizeof(float)*ndims_wid, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void **) &d_mu_wid, sizeof(float)*ndims_wid));
	checkCudaErrors(hipMemcpy(d_mu_wid, &(mu_wid[0]), sizeof(float)*ndims_wid, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**) &d_mu_wid_weighted, sizeof(float)*ndims_wid));
	checkCudaErrors(hipMalloc((void**) &d_Wid, sizeof(float)*ndims_wid));

	checkCudaErrors(hipMalloc((void **) &d_sigma_wid, sizeof(float)*ndims_wid*ndims_wid));
	checkCudaErrors(hipMemcpy(d_sigma_wid, &(sigma_wid[0]), sizeof(float)*ndims_wid*ndims_wid, hipMemcpyHostToDevice));

	// write back for examiniation
	PhGUtils::write2file(sigma_wid, "wid.txt");

	PhGUtils::message("done.");
	showCUDAMemoryUsage();

	// expression prior
	PhGUtils::message("Loading expression prior data ...");
	const string fnwexp = "../Data/blendshape/wexp.bin";
	ifstream fwexp(fnwexp, ios::in | ios::binary );

	fwexp.read(reinterpret_cast<char*>(&ndims_wexp), sizeof(int));
	cout << "expression prior dim = " << ndims_wexp << endl;
	vector<float> mu_wexp0, mu_wexp, sigma_wexp;
	mu_wexp0.resize(ndims_wexp);
	mu_wexp.resize(ndims_wexp);
	sigma_wexp.resize(ndims_wexp*ndims_wexp);

	fwexp.read(reinterpret_cast<char*>(&(mu_wexp0[0])), sizeof(float)*ndims_wexp);
	fwexp.read(reinterpret_cast<char*>(&(mu_wexp[0])), sizeof(float)*ndims_wexp);
	fwexp.read(reinterpret_cast<char*>(&(sigma_wexp[0])), sizeof(float)*ndims_wexp*ndims_wexp);

	fwexp.close();

	PhGUtils::message("expression prior loaded.");
	PhGUtils::message("transferring expression prior to GPU ...");

	// transfer the expression prior to GPU
	checkCudaErrors(hipMalloc((void **) &d_mu_wexp0, sizeof(float)*ndims_wexp));
	checkCudaErrors(hipMemcpy(d_mu_wexp0, &(mu_wexp0[0]), sizeof(float)*ndims_wexp, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void **) &d_mu_wexp, sizeof(float)*ndims_wexp));
	checkCudaErrors(hipMemcpy(d_mu_wexp, &(mu_wexp[0]), sizeof(float)*ndims_wexp, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**) &d_mu_wexp_weighted, sizeof(float)*ndims_wexp));
	checkCudaErrors(hipMalloc((void**) &d_Wexp, sizeof(float)*ndims_wexp));

	checkCudaErrors(hipMalloc((void **) &d_sigma_wexp, sizeof(float)*ndims_wexp*ndims_wexp));
	checkCudaErrors(hipMemcpy(d_sigma_wexp, &(sigma_wexp[0]), sizeof(float)*ndims_wexp*ndims_wexp, hipMemcpyHostToDevice));

	// write back for examination
	PhGUtils::write2file(sigma_wexp, "wexp.txt");

	PhGUtils::message("done.");
	showCUDAMemoryUsage();

	// load the indices of landmarks
	const string lmfn = "../Data/model/landmarks.txt";
	ifstream fin(lmfn, ios::in);
	if( fin.is_open() ) {
		landmarkIdx.reserve(128);
		int idx;
		while(fin.good()) {
			fin >> idx;
			landmarkIdx.push_back(idx);
		}
		PhGUtils::message("landmarks loaded.");
		cout << "total landmarks = " << landmarkIdx.size() << endl;
		ndims_fpts = landmarkIdx.size() * 3;
		nfpts = landmarkIdx.size();
	}
	else {
		PhGUtils::abort("Failed to load landmarks!");
	}
	// allocate space for landmarks
	checkCudaErrors(hipMalloc((void**) &d_fptsIdx, sizeof(int)*landmarkIdx.size()));
	// upload the landmark indices
	checkCudaErrors(hipMemcpy(d_fptsIdx, &(landmarkIdx[0]), sizeof(int)*landmarkIdx.size(), hipMemcpyHostToDevice));

	/*
	h_q = new float[landmarkIdx.size()*3];
	checkCudaErrors(hipMalloc((void**) &d_q, sizeof(float)*landmarkIdx.size()*3));
	h_q2d = new float[landmarkIdx.size()*3];
	checkCudaErrors(hipMalloc((void**) &d_q2d, sizeof(float)*landmarkIdx.size()*3));
	*/
	checkCudaErrors(hipHostAlloc((void**) &h_q, sizeof(float)*landmarkIdx.size()*3, hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_q, h_q, 0));
	checkCudaErrors(hipHostAlloc((void**) &h_q2d, sizeof(float)*landmarkIdx.size()*3, hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_q2d, h_q2d, 0));

	ndims_pts = core_dim[2];	// constraints by the vertices, at most 3 constraints for each vertex

	checkCudaErrors(hipMalloc((void**) &d_targetLocations, sizeof(float)*ndims_pts));
	showCUDAMemoryUsage();

	PhGUtils::message("allocating memory for computataion ...");
	// allocate space for Aid, Aexp, AidtAid, AexptAexp, brhs, Aidtb, Aexptb
	checkCudaErrors(hipMalloc((void **) &d_RTparams, sizeof(float)*7));
	checkCudaErrors(hipMalloc((void **) &d_meanRT, sizeof(float)*7));

	int maxParams = max(ndims_wid, ndims_wexp);
	checkCudaErrors(hipMalloc((void **) &d_A, sizeof(double)*(maxParams + ndims_fpts + ndims_pts) * maxParams));
	checkCudaErrors(hipMalloc((void **) &d_b, sizeof(double)*(ndims_pts + ndims_fpts + maxParams)));

	checkCudaErrors(hipMalloc((void **) &d_AtA, sizeof(double)*maxParams*maxParams));
	checkCudaErrors(hipMalloc((void **) &d_Atb, sizeof(double)*maxParams));

	/*
	h_w_mask = new float[landmarkIdx.size()];
	checkCudaErrors(hipMalloc((void**) &d_w_mask, sizeof(float)*landmarkIdx.size()));

	h_w_landmarks = new float[landmarkIdx.size()];
	checkCudaErrors(hipMalloc((void**) &d_w_landmarks, sizeof(float)*landmarkIdx.size()));
	*/
	checkCudaErrors(hipHostAlloc((void**) &h_w_mask, sizeof(float)*landmarkIdx.size(), hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_w_mask, h_w_mask, 0));
	checkCudaErrors(hipHostAlloc((void**) &h_w_landmarks, sizeof(float)*landmarkIdx.size(), hipHostMallocMapped));
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_w_landmarks, h_w_landmarks, 0));


	checkCudaErrors(hipMalloc((void**) &d_icpc, sizeof(d_ICPConstraint)*MAX_ICPC_COUNT));
	checkCudaErrors(hipMalloc((void**) &d_nicpc, sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &d_icpc_rigid, sizeof(d_ICPConstraint)*MAX_ICPC_COUNT));
	checkCudaErrors(hipMalloc((void**) &d_nicpc_rigid, sizeof(int)));


	h_error = new float[MAX_ICPC_COUNT];
	checkCudaErrors(hipMalloc((void**) &d_error, sizeof(float)*MAX_ICPC_COUNT));
	h_w_error = new float[MAX_ICPC_COUNT];
	checkCudaErrors(hipMalloc((void**) &d_w_error, sizeof(float)*MAX_ICPC_COUNT));
	PhGUtils::message("done.");

	PhGUtils::message("allocating memory for incoming data ...");
	checkCudaErrors(hipMalloc((void**) &d_colordata, sizeof(unsigned char)*640*480*4));
	checkCudaErrors(hipMalloc((void**) &d_depthdata, sizeof(unsigned char)*640*480*4));

	checkCudaErrors(hipMalloc((void**) &d_indexMap, sizeof(unsigned char)*640*480*4));
	checkCudaErrors(hipMalloc((void**) &d_depthMap, sizeof(float)*640*480));
	PhGUtils::message("done.");

	showCUDAMemoryUsage();
}

__host__ void MultilinearReconstructorGPU::initializeWeights() {
	// read the weights from a setting file
	string fweights = "../Data/weights.txt";
	ifstream fin(fweights, ios::in);
	if( !fin ){
		PhGUtils::fail("failed to load weights file. using default weights");
		w_prior_id = 1e-3;
		w_prior_exp = 7.5e-4;

		w_boundary = 1e-8;
		w_chin = 5e-6;
		w_outer = 2.5e-10;
		w_fp = 0.25;

		w_history = 0.01;
		w_ICP = 1.0;		
	}
	else {
		fin >> w_prior_id >> w_prior_exp
			>> w_boundary >> w_chin >> w_outer
			>> w_fp >> w_ICP >> w_history;
	}
	fin.close();


	historyWeights[0] = 0.02;
	historyWeights[1] = 0.04;
	historyWeights[2] = 0.08;
	historyWeights[3] = 0.16;
	historyWeights[4] = 0.32;
	historyWeights[5] = 0.64;
	historyWeights[6] = 1.28;
	historyWeights[7] = 2.56;
	historyWeights[8] = 5.12;
	historyWeights[9] = 10.24;

	for(int i=0;i<78;i++) {
		if( i < 42 || i > 74 ) h_w_mask[i] = w_fp;
		else {
			if( i > 63 ) h_w_mask[i] = w_outer;
			else h_w_mask[i] = w_chin;
		}
	}
	//checkCudaErrors(hipMemcpy(d_w_mask, w_mask, sizeof(float)*78, hipMemcpyHostToDevice));

	totalCons = 0;
}

__host__ void MultilinearReconstructorGPU::bindTarget(const vector<PhGUtils::Point3f>& pts)
{
	//cout << "binding " << pts.size() << " targets ..." << endl;
	// update q array and q2d array on host side
	// they are stored in page-locked memory
	int numpts = pts.size();
	for(int i=0;i<numpts;i++) {
		int idx = i*3;
		h_q2d[idx] = pts[i].x, h_q2d[idx+1] = pts[i].y, h_q2d[idx+2] = pts[i].z;
		PhGUtils::colorToWorld(pts[i].x, pts[i].y, pts[i].z, h_q[idx], h_q[idx+1], h_q[idx+2]);
	}

	// compute depth mean and variance
	int validZcount = 0;
	float mu_depth = 0, sigma_depth = 0;
	for(int i=0;i<numpts;i++) {
		float z = h_q[i*3+2];
		if( z != 0 ){
			mu_depth += z;
			validZcount++;
		}
	}
	mu_depth /= validZcount;
	for(int i=0;i<numpts;i++) {
		float z = h_q[i*3+2];
		if( z != 0 ){
			float dz = z - mu_depth;
			sigma_depth += dz * dz;
		}
	}
	sigma_depth /= (validZcount-1);

	const float DEPTH_THRES = 1e-6;
	int validCount = 0;
	meanX = 0; meanY = 0; meanZ = 0;
	// initialize weights
	for(int i=0, idx=0;i<numpts;i++, idx+=3) {
		const float3& p = make_float3(h_q[idx], h_q[idx+1], h_q[idx+2]);
		int isValid = (fabs(p.z) > DEPTH_THRES)?1:0;

		meanX += p.x * isValid;
		meanY += p.y * isValid;
		meanZ += p.z * isValid;

		float dz = p.z - mu_depth;
		float w_depth = exp(-fabs(dz) / (sigma_depth*100.0));

		// set the landmark weights
		h_w_landmarks[i] = (i<64 || i>74)?isValid*w_depth:1.0;
		validCount += isValid;
	}

	// upload to GPU
	/*
	//PhGUtils::message("uploading targets to GPU ...");
	hipMemcpy(d_q2d, h_q2d, sizeof(float)*numpts*3, hipMemcpyHostToDevice);
	checkCudaState();
	//writeback(d_q2d, numpts, 3, "d_q2d.txt");
	hipMemcpy(d_q, h_q, sizeof(float)*numpts*3, hipMemcpyHostToDevice);
	checkCudaState();
	//writeback(d_q, numpts, 3, "d_q.txt");
	hipMemcpy(d_w_landmarks, h_w_landmarks, sizeof(float)*numpts, hipMemcpyHostToDevice);
	checkCudaState();
	//PhGUtils::message("done.");
	*/
}

__host__ void MultilinearReconstructorGPU::bindRGBDTarget(const vector<unsigned char>& colordata,
														  const vector<unsigned char>& depthdata) 
{
	//PhGUtils::message("uploading image targets to GPU ...");

	// update both color data and depth data
	const int sz = sizeof(unsigned char)*640*480*4;
	hipMemcpy(d_colordata, &(colordata[0]), sz, hipMemcpyHostToDevice);
	checkCudaState();
	hipMemcpy(d_depthdata, &(depthdata[0]), sz, hipMemcpyHostToDevice);
	checkCudaState();

	//PhGUtils::message("done.");
}

__host__ void MultilinearReconstructorGPU::setBaseMesh(const PhGUtils::QuadMesh& m) {
	baseMesh = m;
	// upload the mesh topology
	int nfaces = baseMesh.faceCount();
	int nverts = baseMesh.vertCount();
	cout << "setting base mesh: #v = " << nverts << ", #f = " << nfaces << endl;
	validfaces = 0;
	frontFaces.clear();
	vector<int4> topo(nfaces);
	//h_meshtopo.resize(nfaces*2*3);
	isBackFace.resize(nfaces);
	h_meshverts.resize(nfaces*4);
	h_faceidx.resize(nfaces*4);
	for(int i=0;i<nfaces;i++) {
		const PhGUtils::QuadMesh::face_t& f = baseMesh.face(i);
		topo[i] = make_int4(f.x, f.y, f.z, f.w);
		const PhGUtils::QuadMesh::vert_t& v0 = baseMesh.vertex(f.x);
		if( v0.z < -0.75 ) isBackFace[i] = true;
		else isBackFace[i] = false;
		//h_meshtopo[i*6+0] = f.x; h_meshtopo[i*6+1] = f.y; h_meshtopo[i*6+2] = f.z;
		//h_meshtopo[i*6+3] = f.y; h_meshtopo[i*6+4] = f.z; h_meshtopo[i*6+5] = f.w;

		float3 clr;
		PhGUtils::encodeIndex<float>(i, clr.x, clr.y, clr.z);		
		// fill the color array		
#if FAST_RENDER
		if( isBackFace[i] ) continue;
		else frontFaces.push_back(i);
		h_faceidx[validfaces+0] = h_faceidx[validfaces+1] = h_faceidx[validfaces+2] = h_faceidx[validfaces+3] = clr;
        validfaces+=4;
#else
		h_faceidx[f.x] = clr;
		h_faceidx[f.y] = clr;
		h_faceidx[f.z] = clr;
		h_faceidx[f.w] = clr;
#endif
	}

	PhGUtils::message("uploading mesh topology");
	cout << "face count = " << nfaces << endl;
	if( d_meshtopo ) {
		checkCudaErrors(hipFree(d_meshtopo));
	}
	checkCudaErrors(hipMalloc((void**) &d_meshtopo, sizeof(int4)*nfaces));
	checkCudaErrors(hipMemcpy(d_meshtopo, &(topo[0]), sizeof(int4)*nfaces, hipMemcpyHostToDevice));

	if( d_meshverts ) {
		checkCudaErrors(hipFree(d_meshverts));
	}
	checkCudaErrors(hipMalloc((void**) &d_meshverts, sizeof(float3)*nfaces*4));

	showCUDAMemoryUsage();
}

__host__ void MultilinearReconstructorGPU::initRenderer() {
	// off-screen rendering related
	depthMap.resize(640*480);
	indexMap.resize(640*480*4);
	mProj = PhGUtils::KinectColorProjection.transposed();
	mMv = PhGUtils::Matrix4x4f::identity();

	dummyWgt = shared_ptr<QGLWidget>(new QGLWidget());
	dummyWgt->hide();
	dummyWgt->makeCurrent();
	fbo = shared_ptr<QGLFramebufferObject>(new QGLFramebufferObject(640, 480, QGLFramebufferObject::Depth));
	dummyWgt->doneCurrent();
}

__host__ void MultilinearReconstructorGPU::fit(FittingOption op) {
	switch( op ) {
	case FIT_POSE:
		{
			fitPose();
			break;
		}
	case FIT_IDENTITY:
		{

			break;
		}
	case FIT_EXPRESSION:
		{

			break;
		}
	case FIT_POSE_AND_IDENTITY:
		{
			fitPoseAndIdentity();
			break;
		}
	case FIT_POSE_AND_EXPRESSION:
		{
			fitPoseAndExpression();
			break;
		}
	case FIT_ALL:
		{

			break;
		}
	}

	// store pose history
	if( useHistory ) {
		// post process, impose a moving average for pose
		RTHistory.push_back(vector<float>(h_RTparams, h_RTparams+7));
		if( RTHistory.size() > historyLength ) RTHistory.pop_front();
		vector<float> m = computeWeightedMeanPose();

		hipMemcpy(d_meanRT, &(m[0]), sizeof(float)*7, hipMemcpyHostToDevice);
		checkCudaState();
	}
}

__host__ void MultilinearReconstructorGPU::fitPose() {
	//cout << "fitting pose ..." << endl;
	
	// make rotation matrix and translation vector
	//cout << "initial guess ..." << endl;
	//PhGUtils::printArray(h_RTparams, 7);

	cc = 1e-4;
	float errorThreshold_ICP = 1e-5;
	float errorDiffThreshold_ICP = errorThreshold * 1e-4;

	int iters = 0;
	float E0 = 0, E;
	bool converged = false;
	const int MaxIterations = 64;

	int rigidIters;

	while( !converged && iters++<MaxIterations ) {
		transformMesh();
		updateMesh();
		renderMesh();
		nicpc = collectICPConstraints(iters, MaxIterations);
		converged = fitRigidTransformation(true, rigidIters);
		E = computeError();
		//PhGUtils::debug("iters", iters, "Error", E);

		// adaptive threshold
		converged |= E < (errorThreshold_ICP / (nicpc/5000.0));
		converged |= fabs(E - E0) < errorDiffThreshold_ICP;
		E0 = E;
	}

	// use the latest parameters
	transformMesh();
	updateMesh();
}

__global__ void transformTM0_kernel(float *d_tm0RT, mat3 R, int npts, int ndims) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= npts ) return;

	int stride = npts * 3;
	int offset  = tid * 3;
	for(int i=0;i<ndims;++i,offset+=stride) {
		rotate_point(R, d_tm0RT[offset], d_tm0RT[offset+1], d_tm0RT[offset+2]);
	}
}

__host__ void MultilinearReconstructorGPU::transformTM0() {
	hipMemcpy(d_tm0RT, d_tm0, sizeof(float)*ndims_pts*ndims_wexp, hipMemcpyDeviceToDevice);
	checkCudaState();
	// call the transformation kernel
	const int threads = 1024;
	mat3 R = mat3::rotation(h_RTparams[0], h_RTparams[1], h_RTparams[2])*h_RTparams[6];
	transformTM0_kernel<<<(int)(ceil(npts_mesh/(float)threads)), threads>>>(d_tm0RT, R, npts_mesh, ndims_wexp);
	checkCudaState();
	//writeback(d_tm0RT, ndims_wexp, npts_mesh*3, "d_tm0RT.txt");
	//checkCudaState();
}

__global__ void transformTM1_kernel(float *d_tm1RT, mat3 R, int npts, int ndims) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= npts ) return;
	int offset  = tid * 3 * ndims;
	for(int i=0;i<ndims;++i,offset++) {
		rotate_point(R, d_tm1RT[offset], d_tm1RT[offset+ndims], d_tm1RT[offset+ndims*2]);
	}
}

__host__ void MultilinearReconstructorGPU::transformTM1() {
	hipMemcpy(d_tm1RT, d_tm1, sizeof(float)*ndims_pts*ndims_wid, hipMemcpyDeviceToDevice);
	checkCudaState();
	// call the transformation kernel
	const int threads = 1024;
	mat3 R = mat3::rotation(h_RTparams[0], h_RTparams[1], h_RTparams[2])*h_RTparams[6];
	transformTM1_kernel<<<(int)(ceil(npts_mesh/(float)threads)), threads>>>(d_tm1RT, R, npts_mesh, ndims_wid);
	checkCudaState();
	//writeback(d_tm1RT, npts_mesh*3, ndims_wid, "d_tm1RT.txt");
	//checkCudaState();
}

__global__ void fitIdentity_ICPCTerm(d_ICPConstraint *d_icpc, int nicpc, int ndims, int off, int boff,
									 float *d_tm1RT, double *d_A, double *d_b,
									 float3 T, float w_ICP) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nicpc ) return;

	const d_ICPConstraint& icpc = d_icpc[tid];
	const float3& bc = icpc.bcoords;
	int3 vidx = icpc.v * 3 * ndims;
	int ndims2 = ndims*2;

	int offset = tid * 3 * ndims + off;
	for(int i=0;i<ndims;++i) {
		int3 k = vidx + i;
		float3 v0 = make_float3(d_tm1RT[k.x], d_tm1RT[k.x+ndims], d_tm1RT[k.x+ndims2]);
		float3 v1 = make_float3(d_tm1RT[k.y], d_tm1RT[k.y+ndims], d_tm1RT[k.y+ndims2]);
		float3 v2 = make_float3(d_tm1RT[k.z], d_tm1RT[k.z+ndims], d_tm1RT[k.z+ndims2]);
		float3 p = (v0 * bc.x + v1 * bc.y + v2 * bc.z) * w_ICP;

		int j = offset + i;
		d_A[j] = p.x;
		d_A[j+ndims] = p.y;
		d_A[j+ndims2] = p.z;
	}

	const float3& q = icpc.q;
	int boffset = tid * 3 + boff;
	d_b[boffset  ] = (q.x - T.x) * w_ICP;
	d_b[boffset+1] = (q.y - T.y) * w_ICP;
	d_b[boffset+2] = (q.z - T.z) * w_ICP;
}

__global__ void fitIdentity_FeaturePointsTerm(int *d_fptsIdx, float *d_q, int nfpts, int ndims, int off, int boff,
											  float *d_tm1RT, double *d_A, double *d_b,											  
											  float3 T, float *d_w_landmarks, float w_fp_scale) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nfpts ) return;

	int offset = tid * 3 * ndims + off;
	float wpt = d_w_landmarks[tid] * w_fp_scale;
	int voffset = d_fptsIdx[tid] * 3 * ndims;
	int ndims2 = ndims*2;

	for(int i=0;i<ndims;++i) {
		int j = offset+i;
		int k = voffset+i;
		float3 p = make_float3(d_tm1RT[k], d_tm1RT[k+ndims], d_tm1RT[k+ndims2]);	
		d_A[j] = p.x * wpt;
		d_A[j+ndims] = p.y * wpt;
		d_A[j+ndims2] = p.z * wpt;
	}

	int boffset = tid * 3 + boff;
	int qoffset = tid * 3;

	if( d_q[qoffset+2] == 0 ) wpt = 0;

	d_b[boffset  ] = (d_q[qoffset  ] - T.x) * wpt;
	d_b[boffset+1] = (d_q[qoffset+1] - T.y) * wpt;
	d_b[boffset+2] = (d_q[qoffset+2] - T.z) * wpt;
}

__global__ void fitIdentity_PriorTerm(double *d_A, double *d_b, float *d_sigma_wid_weighted, float *d_mu_wid_weighted,
									  int ndims, int off, int boff, 
									  float w_prior_scale) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= ndims ) return;

	int poffset = tid*ndims;
	int offset = poffset+off;	
	for(int i=0;i<ndims;i++) {		
		d_A[offset++] = d_sigma_wid_weighted[poffset++] * w_prior_scale;
	}

	d_b[tid+boff] = d_mu_wid_weighted[tid] * w_prior_scale;
}

__global__ void copySolutionToB(double *d_Atb, float *d_b, int ndims)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= ndims ) return;

	const float alpha = 1.0;
	d_b[tid] = alpha * d_Atb[tid] + (1.0 - alpha) * d_b[tid];
}

__host__ bool MultilinearReconstructorGPU::fitIdentityWeights() {
	float3 T = make_float3(h_RTparams[3], h_RTparams[4], h_RTparams[5]);
	const int threads = 1024;
	checkCudaState();
	// assemble the matrix and right hand side
	fitIdentity_ICPCTerm<<<(int)(ceil(nicpc/(float)threads)),threads>>>(d_icpc, nicpc, ndims_wid, 0, 0,
									 d_tm1RT, d_A, d_b,
									 T, w_ICP);
	checkCudaState();
	//cout << nicpc << ", " << ndims_wid << endl;
	//writeback(d_A, nicpc*3, ndims_wid, "d_A0.txt");
	//writeback(d_b, nicpc*3, 1, "d_b0.txt");
	checkCudaState();
	fitIdentity_FeaturePointsTerm<<<1, 128>>>(d_fptsIdx, d_q, nfpts, ndims_wid, nicpc*ndims_wid*3, nicpc*3,
											  d_tm1RT, d_A, d_b,											  
											  T, d_w_landmarks, w_fp);
	checkCudaState();
	//writeback(d_A, (nicpc+nfpts)*3, ndims_wid, "d_A1.txt");
	//writeback(d_b, (nicpc+nfpts)*3, 1, "d_b1.txt");
	checkCudaState();
	fitIdentity_PriorTerm<<<1, 64>>>(d_A, d_b, d_sigma_wid, d_mu_wid_weighted,
									  ndims_wid, (nicpc+nfpts)*ndims_wid*3, (nicpc+nfpts)*3, 
									  w_prior_id);
	checkCudaState();
	//writeback(d_A, (nicpc+nfpts)*3+ndims_wid, ndims_wid, "d_A2.txt");
	//writeback(d_b, (nicpc+nfpts)*3+ndims_wid, 1, "d_b2.txt");
	checkCudaState();

	vector<float> Wid(ndims_wid), brhs(ndims_wid);
	hipMemcpy(&Wid[0], d_Wid, sizeof(float)*ndims_wid, hipMemcpyDeviceToHost);
	checkCudaState();

	// solve for new set of parameters
#if USE_LS_SOLVER
	culaStatus s = culaDeviceSgels('T', ndims_wid, (nicpc+nfpts)*3+ndims_wid, 1, d_A, ndims_wid, d_b, (nicpc+nfpts)*3+ndims_wid);
	if( s != culaNoError ) {
		cerr << "cula failed!" << endl;
		if( s == culaArgumentError )
			printf("Argument %d has an illegal value\n", culaGetErrorInfo());
		else if( s == culaDataError )
			printf("Data error with code %d, please see LAPACK documentation\n", culaGetErrorInfo());
		else
			printf("%s\n", culaGetStatusString(s));		
	}

	hipMemcpy(d_Wid, d_b, sizeof(float)*ndims_wid, hipMemcpyDeviceToDevice);
	checkCudaState();
#else
	// normal mat method
	hipblasDsyrk('U', 'N', ndims_wid, (nicpc+nfpts)*3+ndims_wid, 1.0, d_A, ndims_wid, 0.0, d_AtA, ndims_wid);
	//writeback(d_AtA, ndims_wid, ndims_wid, "d_AtA.txt");
	hipblasDgemv('N', ndims_wid, (nicpc+nfpts)*3+ndims_wid, 1.0, d_A, ndims_wid, d_b, 1, 0.0, d_Atb, 1);
	//writeback(d_Atb, ndims_wid, 1, "d_Atb.txt");

	culaDeviceDpotrf('U', ndims_wid, d_AtA, ndims_wid);
	culaDeviceDpotrs('U', ndims_wid, 1, d_AtA, ndims_wid, d_Atb, ndims_wid);

	copySolutionToB<<<1, 64>>>(d_Atb, d_Wid, ndims_wid);
	//writeback(d_Wid, ndims_wid, 1, "d_wid.txt");
	checkCudaState();
#endif

	hipMemcpy(&brhs[0], d_b, sizeof(float)*ndims_wid, hipMemcpyDeviceToHost);
	checkCudaState();

	float diff = 0;
	for(int i=0;i<ndims_wid;i++) {
		diff += fabs(Wid[i] - brhs[i]);
		//cout << Wid[i] << "\t" << brhs[i] << endl;
	}
	//cout << endl;

	return diff/ndims_wid < cc;
}

__host__ void MultilinearReconstructorGPU::fitPoseAndIdentity() {
	cc = 1e-4;
	float errorThreshold_ICP = 1e-5;
	float errorDiffThreshold_ICP = errorThreshold * 1e-4;

	int iters = 0;
	float E0 = 0, E;
	bool converged = false;
	const int MaxIterations = 64;
	int rigidIters;

	while( !converged && iters++<MaxIterations ) {
		converged = true;
		transformMesh();
		updateMesh();
		renderMesh();
		nicpc = collectICPConstraints(iters, MaxIterations);
		converged &= fitRigidTransformation(true, rigidIters);

		// transform tm1
		transformTM1();

		// fit identity weights
		converged &= fitIdentityWeights();
		
		// update tplt with tm1
		hipblasSgemv('T', ndims_wid, ndims_pts, 1.0, d_tm1, ndims_wid, d_Wid, 1, 0.0, d_tplt, 1);
		//writeback(d_tplt, npts_mesh, 3, "d_tplt.txt");

		// update distance map
		hipblasSgemv('N', npts_mesh, ndims_wid, 1.0, d_rawdistmap, npts_mesh, d_Wid, 1, 0, d_distmap, 1);
		checkCudaState();
		//writeback(d_distmap, npts_mesh, "d_distmap.txt");
		//writeback(d_rawdistmap, ndims_wid, npts_mesh, "d_rawdistmap.txt");
		//::system("pause");

		E = computeError();
		PhGUtils::debug("iters", iters, "Error", E, "Error diff", fabs(E-E0));

		// adaptive threshold
		converged |= E < (errorThreshold_ICP / (nicpc/5000.0));
		converged |= fabs(E - E0) < errorDiffThreshold_ICP;
		E0 = E;
	}

	// use the latest parameters
	transformMesh();
	updateMesh();
		
	// update tm0, for the following steps
	hipblasSgemv('N', ndims_wexp * ndims_pts, ndims_wid, 1.0, d_tu0, ndims_wexp * ndims_pts, d_Wid, 1, 0, d_tm0, 1);

	// update distance map
	hipblasSgemv('N', npts_mesh, ndims_wid, 1.0, d_rawdistmap, npts_mesh, d_Wid, 1, 0, d_distmap, 1);
}

__global__ void fitExpression_ICPCTerm(d_ICPConstraint *d_icpc, int nicpc, int npts, int ndims, int off, int boff,
									 float *d_tm1RT, double *d_A, double *d_b,
									 float3 T, float w_ICP) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nicpc ) return;

	const d_ICPConstraint& icpc = d_icpc[tid];
	const float3& bc = icpc.bcoords;
	int3 vidx = icpc.v * 3;

	int offset = tid * 3 * ndims + off;
	int boffset = tid * 3 + boff;
	int stride = npts * 3;
	for(int i=0;i<ndims;++i) {
		int3 j = vidx + i * stride;
		float3 v0 = make_float3(d_tm1RT[j.x], d_tm1RT[j.x+1], d_tm1RT[j.x+2]);
		float3 v1 = make_float3(d_tm1RT[j.y], d_tm1RT[j.y+1], d_tm1RT[j.y+2]);
		float3 v2 = make_float3(d_tm1RT[j.z], d_tm1RT[j.z+1], d_tm1RT[j.z+2]);
		float3 p = (v0 * bc.x + v1 * bc.y + v2 * bc.z) * w_ICP;

		d_A[offset+i] = p.x;
		d_A[offset+i+ndims] = p.y;
		d_A[offset+i+ndims*2] = p.z;
	}

	const float3& q = icpc.q;

	d_b[boffset  ] = (q.x - T.x) * w_ICP;
	d_b[boffset+1] = (q.y - T.y) * w_ICP;
	d_b[boffset+2] = (q.z - T.z) * w_ICP;
}

__global__ void fitExpression_FeaturePointsTerm(int *d_fptsIdx, float *d_q, int nfpts, int npts, int ndims, int off, int boff,
											  float *d_tm1RT, double *d_A, double *d_b,											  
											  float3 T, float *d_w_landmarks, float w_fp_scale) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nfpts ) return;

	int offset = tid * 3 * ndims + off;
	float wpt = d_w_landmarks[tid] * w_fp_scale;
	int voffset = d_fptsIdx[tid] * 3;
	int stride = npts * 3;
	for(int i=0;i<ndims;++i) {
		int j = offset+i;
		int k = voffset+i * stride;
		float3 p = make_float3(d_tm1RT[k], d_tm1RT[k+1], d_tm1RT[k+2]);	
		d_A[j] = p.x * wpt;
		d_A[j+ndims] = p.y * wpt;
		d_A[j+ndims*2] = p.z * wpt;
	}

	int boffset = tid * 3 + boff;
	int qoffset = tid * 3;

	if( d_q[qoffset+2] == 0 ) wpt = 0;

	d_b[boffset  ] = (d_q[qoffset  ] - T.x) * wpt;
	d_b[boffset+1] = (d_q[qoffset+1] - T.y) * wpt;
	d_b[boffset+2] = (d_q[qoffset+2] - T.z) * wpt;
}

__global__ void fitExpression_PriorTerm(double *d_A, double *d_b, float *d_sigma_wid_weighted, float *d_mu_wid_weighted,
									  int ndims, int off, int boff, 
									  float w_prior_scale) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= ndims ) return;

	int poffset = tid*ndims;
	int offset = poffset+off;	
	for(int i=0;i<ndims;i++) {		
		d_A[offset++] = d_sigma_wid_weighted[poffset++] * w_prior_scale;
	}

	d_b[tid+boff] = d_mu_wid_weighted[tid] * w_prior_scale;
}

__host__ bool MultilinearReconstructorGPU::fitExpressionWeights() {
	float3 T = make_float3(h_RTparams[3], h_RTparams[4], h_RTparams[5]);
	const int threads = 1024;
	checkCudaState();
	// assemble the matrix and right hand side
	fitExpression_ICPCTerm<<<(int)(ceil(nicpc/(float)threads)),threads>>>(d_icpc, nicpc, npts_mesh, ndims_wexp, 0, 0,
									 d_tm0RT, d_A, d_b,
									 T, w_ICP);
	checkCudaState();
	//cout << nicpc << ", " << ndims_wid << endl;
	//writeback(d_A, nicpc*3, ndims_wexp, "d_Aexp0.txt");
	checkCudaState();
	//cout << w_fp_scale << endl;
	const float w_fp = 5.0;
	fitExpression_FeaturePointsTerm<<<1, 128>>>(d_fptsIdx, d_q, nfpts, npts_mesh, ndims_wexp, nicpc*ndims_wexp*3, nicpc*3,
											  d_tm0RT, d_A, d_b,											  
											  T, d_w_landmarks, w_fp);
	checkCudaState();
	//writeback(d_A, (nicpc+nfpts)*3, ndims_wexp, "d_Aexp1.txt");
	checkCudaState();
	const float w_p = 2.5;
	fitExpression_PriorTerm<<<1, 64>>>(d_A, d_b, d_sigma_wexp, d_mu_wexp_weighted,
									  ndims_wexp, (nicpc+nfpts)*ndims_wexp*3, (nicpc+nfpts)*3, 
									  w_prior_exp);
	checkCudaState();
	//writeback(d_A, (nicpc+nfpts)*3+ndims_wexp, ndims_wexp, "d_Aexp2.txt");
	//checkCudaState();

	vector<float> Wexp(ndims_wexp), brhs(ndims_wexp);
	hipMemcpy(&Wexp[0], d_Wexp, sizeof(float)*ndims_wexp, hipMemcpyDeviceToHost);
	checkCudaState();

#if USE_LS_SOLVER
	// solve for new set of parameters
	culaStatus s = culaDeviceSgels('T', ndims_wexp, (nicpc+nfpts)*3+ndims_wexp, 1, d_A, ndims_wexp, d_b, (nicpc+nfpts)*3+ndims_wexp);
	if( s != culaNoError ) {
		cerr << "cula failed!" << endl;
		if( s == culaArgumentError )
			printf("Argument %d has an illegal value\n", culaGetErrorInfo());
		else if( s == culaDataError )
			printf("Data error with code %d, please see LAPACK documentation\n", culaGetErrorInfo());
		else
			printf("%s\n", culaGetStatusString(s));
	}

	hipMemcpy(d_Wexp, d_b, sizeof(float)*ndims_wexp, hipMemcpyDeviceToDevice);
	checkCudaState();

#else
	// normal mat method
	hipblasDsyrk('U', 'N', ndims_wexp, (nicpc+nfpts)*3+ndims_wexp, 1.0, d_A, ndims_wexp, 0.0, d_AtA, ndims_wexp);
	//writeback(d_AtA, ndims_wexp, ndims_wexp, "d_AtA.txt");
	hipblasDgemv('N', ndims_wexp, (nicpc+nfpts)*3+ndims_wexp, 1.0, d_A, ndims_wexp, d_b, 1, 0.0, d_Atb, 1);
	//writeback(d_Atb, ndims_wexp, 1, "d_Atb.txt");

	culaDeviceDpotrf('U', ndims_wexp, d_AtA, ndims_wexp);
	culaDeviceDpotrs('U', ndims_wexp, 1, d_AtA, ndims_wexp, d_Atb, ndims_wexp);

	copySolutionToB<<<1, 64>>>(d_Atb, d_Wexp, ndims_wexp);
	//writeback(d_Wexp, ndims_wexp, 1, "d_wexp.txt");
	checkCudaState();
#endif

	// full update
	hipMemcpy(&brhs[0], d_b, sizeof(float)*ndims_wexp, hipMemcpyDeviceToHost);
	checkCudaState();

	float diff = 0;
	//b.print("b");
	for(int i=0;i<ndims_wexp;i++) {
		diff += fabs(Wexp[i] - brhs[i]);
		//cout << '#' << i << ": " << Wexp[i] << '\t' << brhs[i] << endl;
	}

	return diff/ndims_wexp < cc;
}

__host__ void MultilinearReconstructorGPU::fitPoseAndExpression() {
	cc = 1e-4;
	float errorThreshold_ICP = 1e-5;
	float errorDiffThreshold_ICP = 1e-4;

	int iters = 0;
	float E0 = 1, E = 0;
	bool converged = false;
	const int MaxIterations = 16;

	constraintCount.clear();
	int rigidIters;

	while( !converged && iters++<MaxIterations ) {
		converged = true;
		tTrans.tic();
		transformMesh();
		tTrans.toc();

		tUpdate.tic();
		updateMesh();
		tUpdate.toc();

		tRender.tic();
		renderMesh();
		tRender.toc();

		tCollect.tic();
		nicpc = collectICPConstraints(iters, MaxIterations);
		constraintCount.push_back(nicpc);
		tCollect.toc();

		tRigid.tic();
		converged &= fitRigidTransformation(false, rigidIters);
		rigidIterations.push_back(rigidIters);
		tRigid.toc();

		tTrans0.tic();
		transformTM0();
		tTrans0.toc();

		tExpr.tic();
		converged &= fitExpressionWeights();
		tExpr.toc();

		tUpdate0.tic();
		hipblasSgemv('N', ndims_pts, ndims_wexp, 1.0, d_tm0, ndims_pts, d_Wexp, 1, 0.0, d_tplt, 1);
		tUpdate0.toc();
		
		//::system("pause");
		tError.tic();
		E = computeError();
		tError.toc();
		//PhGUtils::debug("iters", iters, "Error", E, "Error diff", fabs((E-E0)/E0));

		// adaptive threshold
		converged |= E < (errorThreshold_ICP / (nicpc/5000.0));
		converged |= fabs((E-E0)/E0) < errorDiffThreshold_ICP;
		E0 = E;
	}

	/*
	ofstream fout;
	fout.open("d_wexp.txt", ios::app);
	vector<float> Wexp(ndims_wexp);
	hipMemcpy(&Wexp[0], d_Wexp, sizeof(float)*ndims_wexp, hipMemcpyDeviceToHost);
	for(int i=0;i<ndims_wexp;i++) 
		fout << Wexp[i] << ' ';
	fout << endl;
	fout.close();
	*/

	/*
	for(int i=0;i<7;i++)
		cout << h_RTparams[i] << '\t';
	cout << endl;
	*/
	double avgcons = PhGUtils::average(constraintCount);
	double avgiters = PhGUtils::average(rigidIterations);
	printf("Finished in %d iterations. \n \
		Average constaints = %8.1f. \n \
		Averate iterations for rigid transformation = %8.1f", iters, avgcons, avgiters);
	totalCons += avgcons;
	totalRigidIters += avgiters;
	constraintCount.clear();
	// use the latest parameters
	transformMesh();
	updateMesh();
}

__host__ void MultilinearReconstructorGPU::fitAll() {
	throw "Not implemented yet";
	cc = 1e-4;
	float errorThreshold_ICP = 1e-5;
	float errorDiffThreshold_ICP = errorThreshold * 1e-4;

	int iters = 0;
	float E0 = 0, E;
	bool converged = false;
	const int MaxIterations = 64;

	int rigidIters;

	while( !converged && iters++<MaxIterations ) {
		transformMesh();
		updateMesh();
		renderMesh();
		nicpc = collectICPConstraints(iters, MaxIterations);
		converged = fitRigidTransformation(true, rigidIters);
		E = computeError();
		//PhGUtils::debug("iters", iters, "Error", E);

		// adaptive threshold
		converged |= E < (errorThreshold_ICP / (nicpc/5000.0));
		converged |= fabs(E - E0) < errorDiffThreshold_ICP;
		E0 = E;
	}

	// use the latest parameters
	transformMesh();
	updateMesh();
}

__host__ void MultilinearReconstructorGPU::renderMesh()
{
	dummyWgt->makeCurrent();
	fbo->bind();

#if FBO_DEBUG_GPU
	cout << (fbo->isBound()?"bounded.":"not bounded.") << endl;
	cout << (fbo->isValid()?"valid.":"invalid.") << endl;
#endif

	glEnable(GL_DEPTH_TEST);
	glDepthMask(GL_TRUE);

	glEnable(GL_CULL_FACE);
	glCullFace(GL_BACK);

	glPushMatrix();

	// setup viewing parameters
	glViewport(0, 0, 640, 480);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glMultMatrixf(mProj.data());

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glMultMatrixf(mMv.data());

	glClearColor(0, 0, 0, 1);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT );

	glShadeModel(GL_SMOOTH);

#if !FAST_RENDER
	baseMesh.drawFaceIndices();
#else
	//Enable the vertex array functionality:
	glEnableClientState(GL_VERTEX_ARRAY);
	//Enable the color array functionality (so we can specify a color for each vertex)
	glEnableClientState(GL_COLOR_ARRAY);
	//pass the vertex pointer:
	glVertexPointer( 3,   //3 components per vertex (x,y,z)
					 GL_FLOAT,
					 sizeof(float3),
					 &h_meshverts[0]);
	//pass the color pointer
	glColorPointer(  3,   //3 components per vertex (r,g,b)
					 GL_FLOAT,
					 sizeof(float3),
					 &h_faceidx[0]);  //Pointer to the first color
	//cout << h_meshtopo.size() / 3 << endl;
	glDrawArrays( GL_QUADS, 0, validfaces );
	//glDrawElements(GL_TRIANGLES, h_meshtopo.size(), GL_UNSIGNED_INT, &h_meshtopo[0]);

	glDisableClientState(GL_VERTEX_ARRAY);
	glDisableClientState(GL_COLOR_ARRAY);
#endif

	glReadPixels(0, 0, 640, 480, GL_DEPTH_COMPONENT, GL_FLOAT, &(depthMap[0]));
#if FBO_DEBUG_GPU
	GLenum errcode = glGetError();
	if (errcode != GL_NO_ERROR) {
		const GLubyte *errString = gluErrorString(errcode);
		fprintf (stderr, "OpenGL Error: %s\n", errString);
	}
#endif

	glReadPixels(0, 0, 640, 480, GL_RGBA, GL_UNSIGNED_BYTE, &(indexMap[0]));
#if FBO_DEBUG_GPU
	errcode = glGetError();
	if (errcode != GL_NO_ERROR) {
		const GLubyte *errString = gluErrorString(errcode);
		fprintf (stderr, "OpenGL Error: %s\n", errString);
	}
#endif

	glPopMatrix();

	glDisable(GL_CULL_FACE);

	fbo->release();
	dummyWgt->doneCurrent();

#if FBO_DEBUG_GPU
	ofstream fout("fbodepth.txt");
	PhGUtils::print2DArray(&(depthMap[0]), 480, 640, fout);
	fout.close();

	QImage img = PhGUtils::toQImage(&(indexMap[0]), 640, 480);	
	img.save("fbo.png");
	::system("pause");
#endif

	// upload result to GPU
	checkCudaErrors(hipMemcpy(d_indexMap, &indexMap[0], sizeof(unsigned char)*640*480*4, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_depthMap, &depthMap[0], sizeof(float)*640*480, hipMemcpyHostToDevice));
}

__global__ void clearICPConstraints(int* nicpc, int *nicpc_rigid) {
	*nicpc = 0;
	*nicpc_rigid = 0;
}

//@note	need to upload the topology of the template mesh for constraint collection
__global__ void collectICPConstraints_kernel(
						float*				mesh,
						int4*				meshtopo,
						float*				d_distmap,
						unsigned char*		indexMap,			// synthesized data
						float*				depthMap,			// synthesized data
						unsigned char*		colordata,			// capture data
						unsigned char*		depthdata,			// capture data
						d_ICPConstraint*	icpc,				// ICP constraints
						int*				nicpc,
						d_ICPConstraint*    icpc_rigid,
						int*				nicpc_rigid,
						float thres
	) {
	float DIST_THRES = thres;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x > 639 || y > 479 ) return;

	int tid = y * 640 + x;

	//if( tid & 0x1 ) return;

	int u = x, v = y;
	int idx = (v * 640 + u)*4;
	int vv = 479 - y;
	int didx = vv * 640 + u;
	
	if( depthMap[didx] < 1.0 ) {
		// valid pixel, see if it is a valid constraint
		float d = (depthdata[idx]<<16|depthdata[idx+1]<<8|depthdata[idx+2]);
		
		// bad pixel
		if( d == 0 ) return;

		// compute target location
		float3 q = color2world_fast(u, v, d);

		// take a small window
		const int wSize = 5;
		//int checkedFaces[9];
		//int checkedCount = 0;
		float closestDist = FLT_MAX;
		int3 closestVerts;
		float3 closestHit;

		// check for the closest point face
		for(int r = max(v - wSize, 0); r <= min(v + wSize, 479); r++) {
			int rr = 479 - r;
			for(int c = max(u - wSize, 0); c <= min(u + wSize, 639); c++) {
				int pidx = rr * 640 + c;
				int poffset = pidx << 2;

				float depthVal = depthMap[pidx];
				if( depthVal < 1.0 ) {
					int fidx = decodeIndex(indexMap[poffset], indexMap[poffset+1], indexMap[poffset+2]);
					fidx = clamp(fidx, 0, 11399);
					//bool checked = false;
					//// see if this face is already checked
					//for(int j=0;j<checkedCount;j++) {
					//	if( fidx == checkedFaces[j] ){
					//		checked = true;
					//		break;
					//	}
					//}
					//if( checked ) continue;
					//else {
					//	checkedFaces[checkedCount] = fidx;
					//	checkedCount++;
					//}


					// not checked yet, check out this face
					int4 f = meshtopo[fidx];
					int4 vidx = f * 3;
					
					float3 v0 = make_float3(mesh[vidx.x], mesh[vidx.x+1], mesh[vidx.x+2]);
					float3 v1 = make_float3(mesh[vidx.y], mesh[vidx.y+1], mesh[vidx.y+2]);
					float3 v2 = make_float3(mesh[vidx.z], mesh[vidx.z+1], mesh[vidx.z+2]);
					float3 v3 = make_float3(mesh[vidx.w], mesh[vidx.w+1], mesh[vidx.w+2]);

					float3 hit1, hit2;
					float dist1 = point_to_triangle_distance(q, v0, v1, v2, hit1);
					float dist2 = point_to_triangle_distance(q, v1, v2, v3, hit2);
				
					// take the smaller one
					if( dist1 < dist2 && dist1 < closestDist) {
						closestDist = dist1;
						closestVerts.x = f.x, closestVerts.y = f.y, closestVerts.z = f.z;
						closestHit = hit1;
					}
					else if( dist2 < closestDist ) {
						closestDist = dist2;
						closestVerts.x = f.y, closestVerts.y = f.z, closestVerts.z = f.w;
						closestHit = hit2;
					}
				}
			}
		}

		if( closestDist < DIST_THRES ) {
			d_ICPConstraint cc;
			cc.q = q;
			cc.v = closestVerts;
			int3 vidx = cc.v*3;
						
			float3 v0 = make_float3(mesh[vidx.x], mesh[vidx.x+1], mesh[vidx.x+2]);
			float3 v1 = make_float3(mesh[vidx.y], mesh[vidx.y+1], mesh[vidx.y+2]);
			float3 v2 = make_float3(mesh[vidx.z], mesh[vidx.z+1], mesh[vidx.z+2]);
			
			cc.bcoords = compute_barycentric_coordinates( closestHit, v0, v1, v2 );
			cc.weight = mean(cc.bcoords * make_float3(d_distmap[cc.v.x], d_distmap[cc.v.y], d_distmap[cc.v.z]));

			int slot = atomicAdd(nicpc, 1);
			__threadfence();
			icpc[slot] = cc;

			const float cutoff = 1e-3;
			if( cc.weight < cutoff ) {
				int slot_rigid = atomicAdd(nicpc_rigid, 1);
				__threadfence();
				icpc_rigid[slot_rigid] = cc;
			}
		}
	}
}

__host__ int MultilinearReconstructorGPU::collectICPConstraints(int iters, int maxIters) {
	const float DIST_THRES_MAX = 0.010;
	const float DIST_THRES_MIN = 0.001;
	float DIST_THRES = DIST_THRES_MAX + (DIST_THRES_MIN - DIST_THRES_MAX) * iters / (float)maxIters;
	//PhGUtils::message("Collecting ICP constraints...");
	
	//writeback(d_depthMap, 480, 640, "d_depthmap.txt");

	clearICPConstraints<<<1, 1, 0, mystream>>>(d_nicpc, d_nicpc_rigid);
	checkCudaState();
	PhGUtils::Timer ticpc;
	//ticpc.tic();
	dim3 block(8, 8, 1);
	dim3 grid(640/block.x, 480/block.y, 1);
	collectICPConstraints_kernel<<<grid, block, 0, mystream>>>( d_mesh,
																d_meshtopo,
																d_distmap,
																d_indexMap,
																d_depthMap,
																d_colordata,
																d_depthdata,
																d_icpc,
																d_nicpc,
																d_icpc_rigid,
																d_nicpc_rigid,
																DIST_THRES);

	hipDeviceSynchronize();
	//ticpc.toc("ICPC collection");
	checkCudaState();
	//PhGUtils::message("ICPC computed.");
	// copy back the number of ICP constraints
	int icpcCount = 0;
	hipMemcpy(&icpcCount, d_nicpc, sizeof(int), hipMemcpyDeviceToHost);
	checkCudaState();
	//cout << "ICPC = " << icpcCount << endl;

	hipMemcpy(&nicpc_rigid, d_nicpc_rigid, sizeof(int), hipMemcpyDeviceToHost);
	checkCudaState();
	//cout << "ICPC_rigid = " << nicpc_rigid << endl;

#if OUTPUT_ICPC
	vector<d_ICPConstraint> icpc(640*480);
	checkCudaErrors(hipMemcpy(&icpc[0], d_icpc, sizeof(d_ICPConstraint)*MAX_ICPC_COUNT, hipMemcpyDeviceToHost));
	ofstream fout("d_icpc.txt");
	for(int i=0;i<icpcCount;i++) {
		float3 bc = icpc[i].bcoords;
		int3 vidx = icpc[i].v * 3;
		float3 p;
		p.x = tmesh(vidx.x  ) * bc.x + tmesh(vidx.y  ) * bc.y + tmesh(vidx.z  ) * bc.z;
		p.y = tmesh(vidx.x+1) * bc.x + tmesh(vidx.y+1) * bc.y + tmesh(vidx.z+1) * bc.z;
		p.z = tmesh(vidx.x+2) * bc.x + tmesh(vidx.y+2) * bc.y + tmesh(vidx.z+2) * bc.z;
		fout << icpc[i].q.x << ' '
			 << icpc[i].q.y << ' '
			 << icpc[i].q.z << ' '
			 << p.x << ' '
			 << p.y << ' '
			 << p.z << ' '
			 << bc.x << ' '
			 << bc.y << ' '
			 << bc.z << ' '
			 << icpc[i].weight
			 << endl;
	}
	fout.close();
	::system("pause");
#endif

	return icpcCount;
}

__host__ vector<float> MultilinearReconstructorGPU::computeWeightedMeanPose() {
	vector<float> m(7, 0);

	float wsum = 0;
	int i=0;
	for(auto it=RTHistory.begin(); it!= RTHistory.end(); ++it) {
		for(int j=0;j<7;j++) {
			m[j] += (*it)[j] * historyWeights[i];
		}
		wsum += historyWeights[i];
		i++;
	}

	for(int j=0;j<7;j++) m[j] /= wsum;
	return m;
}

__host__ bool MultilinearReconstructorGPU::fitRigidTransformation(bool fitScale, int& iters) {
	int nparams = fitScale?7:6;

	d_ICPConstraint *icpc_ptr;
	int icpccount;

	if( fitScale ) {
		icpc_ptr = d_icpc;
		icpccount = nicpc;
	}
	else {
		icpc_ptr = d_icpc_rigid;
		icpccount = nicpc_rigid;
	}

	hipMemcpy(NumericalAlgorithms::x, d_RTparams, sizeof(float)*7, hipMemcpyDeviceToDevice);
	checkCudaState();
	int itmax = 32;
	float opts[] = {0.125, 1e-6, 1e-4};
	// gauss-newton algorithm to estimate a new set of parameters
	iters = NumericalAlgorithms::lm(
		nparams, nfpts+nicpc_rigid, itmax, opts,
		d_fptsIdx, d_q, d_q2d, nfpts, d_w_landmarks, d_w_mask, w_fp,
		icpc_ptr, icpccount, w_ICP,
		d_meanRT, w_history,
		d_tplt,
		mystream
		);
	hipDeviceSynchronize();
	PhGUtils::message("rigid transformation estimated in " + PhGUtils::toString(iters) + " iterations.");
	// update the parameters and check if convergence is obtained
	hipMemcpy(d_RTparams, NumericalAlgorithms::x, sizeof(float)*nparams, hipMemcpyDeviceToDevice);
	checkCudaState();
	vector<float> RTparams(nparams);
	hipMemcpy(&(RTparams[0]), NumericalAlgorithms::x, sizeof(float)*nparams, hipMemcpyDeviceToHost);
	checkCudaState();

	//PhGUtils::message("gauss-newton returned.");
	float diff = 0;
	for(int i=0;i<nparams;i++) {
		diff += fabs(RTparams[i] - h_RTparams[i]);
		h_RTparams[i] = RTparams[i];
		//cout << RTparams[i] << ' ';
	}
	//cout << endl;
	//::system("pause");

	return diff/nparams<cc || iters == 0;
}

__global__ void computeError_ICP(float *params, float *d_error, float *d_w_error, int offset, 
								 d_ICPConstraint *d_icpc, int nicpc, float w_ICP, float *d_tplt) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nicpc ) return;

	float s, rx, ry, rz, tx, ty, tz;
	rx = params[0], ry = params[1], rz = params[2];
	tx = params[3], ty = params[4], tz = params[5];
	s = params[6];

	mat3 R = mat3::rotation(rx, ry, rz) * s;
	float3 T = make_float3(tx, ty, tz);

	int3 v = d_icpc[tid].v;
	float3 bc = d_icpc[tid].bcoords;

	int3 vidx = v * 3;

	float3 v0 = make_float3(d_tplt[vidx.x], d_tplt[vidx.x+1], d_tplt[vidx.x+2]);
	float3 v1 = make_float3(d_tplt[vidx.y], d_tplt[vidx.y+1], d_tplt[vidx.y+2]);
	float3 v2 = make_float3(d_tplt[vidx.z], d_tplt[vidx.z+1], d_tplt[vidx.z+2]);
	float3 p = v0 * bc.x + v1 * bc.y + v2 * bc.z;

	const float3& q = d_icpc[tid].q;

	// p = R * p + T
	p = R * p + T;

	d_w_error[tid+offset] = w_ICP;
	d_error[tid+offset] = dot(p-q, p-q) * w_ICP;
}

__global__ void computeError_FeaturePoints(float *params, float *d_error, float *d_w_error, int offset,
		int *d_fptsIdx, float *d_q, float *d_q2d, int nfpts,
		float *d_tplt,
		float *d_w_landmarks, float *d_w_mask,
		float w_fp_scale) 
	{
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if( tid >= nfpts ) return;

		float s, rx, ry, rz, tx, ty, tz;
		rx = params[0], ry = params[1], rz = params[2];
		tx = params[3], ty = params[4], tz = params[5];
		s = params[6];

		mat3 R = mat3::rotation(rx, ry, rz) * s;
		float3 T = make_float3(tx, ty, tz);

		int voffset = tid * 3;
		float wpt = d_w_landmarks[tid] * w_fp_scale * d_w_mask[tid];

		int vidx = d_fptsIdx[tid] * 3;
		float3 p = make_float3(d_tplt[vidx], d_tplt[vidx+1], d_tplt[vidx+2]);
		p = R * p + T;


		if( tid < 42 || tid > 74 ) {
			float3 q = make_float3(d_q[voffset], d_q[voffset+1], d_q[voffset+2]);
			d_error[tid+offset] = dot(p-q, p-q)*wpt;
			d_w_error[tid+offset] = wpt;
		}
		else {
			float3 q = make_float3(d_q2d[voffset], d_q2d[voffset+1], d_q2d[voffset+2]);
			float3 uvd = world2color(p);
			float du = uvd.x - q.x, dv = uvd.y - q.y;
			d_error[tid+offset] = (du*du+dv*dv)*wpt;
			d_w_error[tid+offset] = wpt;
		}
	}

__host__ float MultilinearReconstructorGPU::computeError() {
	checkCudaState();
	//cout << d_error << endl;
	//cout << d_w_error << endl;
	computeError_ICP<<<dim3((int)(ceil(nicpc/1024.0)), 1, 1), dim3(1024, 1, 1), 0, mystream>>>(d_RTparams, d_error, d_w_error, 0, d_icpc, nicpc, w_ICP / nicpc, d_tplt);
	checkCudaState();
	computeError_FeaturePoints<<<dim3(1, 1, 1), dim3(nfpts, 1, 1), 0, mystream>>>(d_RTparams, d_error, d_w_error, nicpc, d_fptsIdx, d_q, d_q2d, nfpts, d_tplt, d_w_landmarks, d_w_mask, w_fp);
	checkCudaState();

	hipMemcpy(h_error, d_error, sizeof(float)*(nicpc+nfpts), hipMemcpyDeviceToHost);
	checkCudaState();
	//writeback(d_error, nicpc+nfpts, 1, "d_error.txt");
	hipMemcpy(h_w_error, d_w_error, sizeof(float)*(nicpc+nfpts), hipMemcpyDeviceToHost);
	checkCudaState();
	float E = 0, Wsum = 0;
	for(int i=0;i<nfpts+nicpc;++i) { E += h_error[i]; Wsum += h_w_error[i]; }

	return E / Wsum;
}

// one dimensional configuration
// R is the rows of the rotation matrix
__device__ float3 R0, R1, R2, T;
__global__ void setupRigidTransformation(float r00, float r01, float r02,
										 float r10, float r11, float r12,
										 float r20, float r21, float r22,
										 float  t0, float  t1,  float t2)
{
	R0 = make_float3(r00, r01, r02);
	R1 = make_float3(r10, r11, r12);
	R2 = make_float3(r20, r21, r22);
	 T = make_float3( t0,  t1,  t2);

	//printf("%f, %f, %f\n", R0.x, R0.y, R0.z);
	//printf("%f, %f, %f\n", R1.x, R1.y, R1.z);
	//printf("%f, %f, %f\n", R2.x, R2.y, R2.z);
}

__global__ void transformMesh_kernel(int nverts, float *d_tplt, float *d_mesh) 
{
	unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if( tid >= nverts ) return;

	unsigned int idx0 = tid*3;

	float3 p = make_float3(d_tplt[idx0], d_tplt[idx0+1], d_tplt[idx0+2]);

	d_mesh[idx0] = dot(R0, p) + T.x;
	d_mesh[idx0+1] = dot(R1, p) + T.y;
	d_mesh[idx0+2] = dot(R2, p) + T.z;
}

__host__ void MultilinearReconstructorGPU::transformMesh() {
	PhGUtils::Matrix3x3f Rot = PhGUtils::rotationMatrix(h_RTparams[0], h_RTparams[1], h_RTparams[2]) * h_RTparams[6];
	//cout << Rot << endl;
	float3 Tvec = make_float3(h_RTparams[3], h_RTparams[4], h_RTparams[5]);
	//cout << Tvec << endl;
	int npts = ndims_pts/3;
	
	checkCudaState();

	setupRigidTransformation<<<1, 1, 0, mystream>>>(Rot(0, 0), Rot(0, 1), Rot(0, 2), Rot(1, 0), Rot(1, 1), Rot(1, 2),
									   Rot(2, 0), Rot(2, 1), Rot(2, 2),	   Tvec.x,    Tvec.y,    Tvec.z);
	checkCudaState();

	//cout << "npts = " << npts << endl;
	dim3 block(256, 1);
	dim3 grid((int)ceil(npts/(float)(block.x)), 1, 1);
	//cout << "grid: " << grid.x << "x" << grid.y << endl;
	transformMesh_kernel<<<grid, block, 0, mystream>>>(npts, d_tplt, d_mesh);
	
	checkCudaState();
}

__global__ void updateMesh_kernel(float* d_mesh, float3* d_meshverts, int4* d_meshtopo, int nfaces) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if( tid >= nfaces ) return;
	
	const int4& vidx = d_meshtopo[tid] * 3;
	int voffset = tid * 4;
	// fill the vertex array
	d_meshverts[voffset+0] = make_float3(d_mesh[vidx.x], d_mesh[vidx.x+1], d_mesh[vidx.x+2]);
	d_meshverts[voffset+1] = make_float3(d_mesh[vidx.y], d_mesh[vidx.y+1], d_mesh[vidx.y+2]);
	d_meshverts[voffset+2] = make_float3(d_mesh[vidx.z], d_mesh[vidx.z+1], d_mesh[vidx.z+2]);
	d_meshverts[voffset+3] = make_float3(d_mesh[vidx.w], d_mesh[vidx.w+1], d_mesh[vidx.w+2]);
}

__host__ void MultilinearReconstructorGPU::updateMesh()
{
	hipMemcpy(tmesh.rawptr(), d_mesh, sizeof(float)*ndims_pts, hipMemcpyDeviceToHost);
	checkCudaState();

	//#pragma omp parallel for
#if !FAST_RENDER
	//cout << "mesh size = " << tmesh.length() << endl;
	//cout << "device mesh address = " << d_mesh << endl;
	//cout << "bytes to transfer = " << sizeof(float)*ndims_pts << endl;

	//writeback(d_mesh, ndims_pts/3, 3, "d_mesh.txt");
	//writeback(d_tplt, ndims_pts/3, 3, "d_tplt.txt");

	for(int i=0;i<tmesh.length()/3;i++) {
		int idx = i * 3;
		baseMesh.vertex(i).x = tmesh(idx++);
		baseMesh.vertex(i).y = tmesh(idx++);
		baseMesh.vertex(i).z = tmesh(idx);
	}
#else
	for(int i=0, validfaces=0;i<frontFaces.size();i++) {
		const PhGUtils::QuadMesh::face_t& f = baseMesh.face(frontFaces[i]);
		int4 vidx = make_int4(f.x, f.y, f.z, f.w)*3;		
		float3 v0 = make_float3(tmesh(vidx.x), tmesh(vidx.x+1), tmesh(vidx.x+2));
		float3 v1 = make_float3(tmesh(vidx.y), tmesh(vidx.y+1), tmesh(vidx.y+2));
		float3 v2 = make_float3(tmesh(vidx.z), tmesh(vidx.z+1), tmesh(vidx.z+2));
		float3 v3 = make_float3(tmesh(vidx.w), tmesh(vidx.w+1), tmesh(vidx.w+2));
		// fill the vertex array
		h_meshverts[validfaces] = v0;
		h_meshverts[validfaces+1] = v1;
		h_meshverts[validfaces+2] = v2;
		h_meshverts[validfaces+3] = v3;
		validfaces+=4;
	}
	
	/*
	updateMesh_kernel<<<(int)ceil(baseMesh.faceCount()/1024.0), 1024>>>(d_mesh, d_meshverts, d_meshtopo, baseMesh.faceCount());
	checkCudaState();
	hipMemcpy(&h_meshverts[0], d_meshverts, sizeof(float3)*baseMesh.faceCount()*4, hipMemcpyDeviceToHost);
	checkCudaState();
	*/
#endif
	

#if 0
	PhGUtils::OBJWriter writer;
	writer.save(baseMesh, "../Data/tmesh.obj");
#endif
}

__host__ void MultilinearReconstructorGPU::printStats() {
	PhGUtils::message("Time cost for transforming mesh = " + PhGUtils::toString(tTrans.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for updating mesh = " + PhGUtils::toString(tUpdate.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for rendering mesh = " + PhGUtils::toString(tRender.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for collecting constraints = " + PhGUtils::toString(tCollect.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for estimating rigid transformation = " + PhGUtils::toString(tRigid.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for transforming template = " + PhGUtils::toString(tTrans0.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for estimating expression weights = " + PhGUtils::toString(tExpr.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for updating template = " + PhGUtils::toString(tUpdate0.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Time cost for computing error = " + PhGUtils::toString(tError.elapsed()*1000.0) + "ms.");
	PhGUtils::message("Total number of constraints = " + PhGUtils::toString(totalCons) + ".");
	PhGUtils::message("Total number of rigid tranformation iterations = " + PhGUtils::toString(totalRigidIters) + ".");
}