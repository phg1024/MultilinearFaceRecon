#include "hip/hip_runtime.h"
#include "MultilinearReconstructorGPU.cuh"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include "Kinect/KinectUtils.h"
#include "Utils/Timer.h"

#include "Elements_GPU.h"

#define FBO_DEBUG_GPU 0
#define KERNEL_DEBUG 0
#define OUTPUT_ICPC 0

MultilinearReconstructorGPU::MultilinearReconstructorGPU():
	d_tu0(nullptr), d_tu1(nullptr), d_tm0(nullptr), d_tm1(nullptr),
	d_tplt(nullptr), d_mesh(nullptr), d_tm0RT(nullptr), d_tm1RT(nullptr),
	d_fptsIdx(nullptr), d_q2d(nullptr), d_q(nullptr), 
	d_colordata(nullptr), d_depthdata(nullptr),
	d_targetLocations(nullptr), d_RTparams(nullptr),
	d_A(nullptr), d_b(nullptr), d_meshtopo(nullptr)
{
	// set device
	cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
	checkCudaState();

	hipSetDeviceFlags(hipDeviceMapHost);

	PhGUtils::message("initializing CULA ...");
	culaInitialize();
	checkCudaState();
	PhGUtils::message("creating CUDA stream ...");
	hipStreamCreate(&mystream);
	checkCudaState();

	w_prior_id = 1e-3;
	w_prior_exp = 1e-4;
	w_boundary = 1e-6;

	meanX = meanY = meanZ = 0;

	// initialize offscreen renderer
	initRenderer();
	
	// initialize members
	init();

	// process the loaded data
	preprocess();

	hipDeviceSynchronize();
}

MultilinearReconstructorGPU::~MultilinearReconstructorGPU() {
	// release resources
	hipDeviceReset();
}

__host__ void MultilinearReconstructorGPU::setPose(const float* params) {
	for(int i=0;i<7;i++) h_RTparams[i] = params[i];
	checkCudaErrors(hipMemcpy(d_RTparams, params, sizeof(float)*7, hipMemcpyHostToDevice));
}

__host__ void MultilinearReconstructorGPU::setIdentityWeights(const Tensor1<float>& t) {
	// copy to GPU
	checkCudaErrors(hipMemcpy(d_Wid, t.rawptr(), sizeof(float)*ndims_wid, hipMemcpyHostToDevice));
}

__host__ void MultilinearReconstructorGPU::setExpressionWeights(const Tensor1<float>& t) {
	// copy to GPU
	checkCudaErrors(hipMemcpy(d_Wexp, t.rawptr(), sizeof(float)*ndims_wexp, hipMemcpyHostToDevice));
}

__host__ void MultilinearReconstructorGPU::preprocess() {
	PhGUtils::message("preprocessing the input data...");

	// process the identity prior

	// invert sigma_wid
	int* ipiv;
	checkCudaErrors(hipMalloc((void**) &ipiv, sizeof(int)*ndims_wid));
	culaDeviceSgetrf(ndims_wid, ndims_wid, d_sigma_wid, ndims_wid, ipiv);
	culaDeviceSgetri(ndims_wid, d_sigma_wid, ndims_wid, ipiv);
	checkCudaErrors(hipFree(ipiv));

	// multiply inv_sigma_wid to mu_wid
	hipblasSgemv('N', ndims_wid, ndims_wid, 1.0, d_sigma_wid, ndims_wid, d_mu_wid, 1, 0.0, d_mu_wid_weighted, 1); 
	
	// scale inv_sigma_wid by w_prior_id
	hipblasSscal(ndims_wid*ndims_wid, w_prior_id, d_sigma_wid, 1);

	// scale mu_wid by w_prior_id
	hipblasSscal(ndims_wid, w_prior_id, d_mu_wid_weighted, 1);

	// copy back the inverted matrix to check correctness
	writeback(d_sigma_wid, ndims_wid*ndims_wid, "invswid.txt");

	// process the expression prior

	// invert sigma_wexp
	checkCudaErrors(hipMalloc((void**) &ipiv, sizeof(int)*ndims_wexp));
	culaDeviceSgetrf(ndims_wexp, ndims_wexp, d_sigma_wexp, ndims_wexp, ipiv);
	culaDeviceSgetri(ndims_wexp, d_sigma_wexp, ndims_wexp, ipiv);
	checkCudaErrors(hipFree(ipiv));

	// multiply inv_sigma_wexp to mu_wexp
	hipblasSgemv('N', ndims_wexp, ndims_wexp, 1.0, d_sigma_wexp, ndims_wexp, d_mu_wexp, 1, 0, d_mu_wexp_weighted, 1);

	// scale inv_sigma_wexp by w_prior_exp
	hipblasSscal(ndims_wexp*ndims_wexp, w_prior_exp, d_sigma_wexp, 1);

	// scale mu_wexp by w_prior_exp
	hipblasSscal(ndims_wexp, w_prior_exp, d_mu_wexp_weighted, 1); 

	writeback(d_sigma_wexp, ndims_wexp*ndims_wexp, "invswexp.txt");
	PhGUtils::message("done.");

	// initialize Wid and Wexp
	checkCudaErrors(hipMemcpy(d_Wid, d_mu_wid0, sizeof(float)*ndims_wid, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_Wexp, d_mu_wexp0, sizeof(float)*ndims_wexp, hipMemcpyDeviceToDevice));

	// initialize tm0, tm1

	// tm0 = tu0 * Wid, use cublas
	// tu0: ndims_wid * (ndims_wexp * ndims_pts) matrix, each row corresponds to an identity
	//		inside each row, the vertices are arranged by expression
	//		That is, a row in tu0 can be see as a row-major matrix where each row corresponds to an expression
	// tm0: a row-major matrix where each row corresponds to an expression
	hipblasSgemv('N', ndims_wexp * ndims_pts, ndims_wid, 1.0, d_tu0, ndims_wexp * ndims_pts, d_Wid, 1, 0, d_tm0, 1);
	writeback(d_tm0, ndims_wexp, ndims_pts, "tm0.txt");

	// tm1 = tu1 * Wexp, use cublas
	// tu1: ndims_wexp * (ndims_wid * ndims_pts) matrix, each row corresponds to an expression
	//		inside each row, the vertices are arraged using index-major
	//		That is, a row in tu1 can be see as a column-major matrix where each column corresponds to an identity
	// tm1: a column-major matrix where each column corresponds to an identity
	hipblasSgemv('N', ndims_wid * ndims_pts, ndims_wexp, 1.0, d_tu1, ndims_wid * ndims_pts, d_Wexp, 1, 0, d_tm1, 1);
	writeback(d_tm1, ndims_pts, ndims_wid, "tm1.txt");

	// create template mesh
	// tplt = tm1 * Wid, use cublas
	hipblasSgemv('T', ndims_wid, ndims_pts, 1.0, d_tm1, ndims_wid, d_Wid, 1, 0.0, d_tplt, 1);
	writeback(d_tplt, ndims_pts/3, 3, "tplt.txt");
}

__host__ void MultilinearReconstructorGPU::init() {
	showCUDAMemoryUsage();
	// read the core tensor
	PhGUtils::message("Loading core tensor ...");
	const string filename = "../Data/blendshape/core.bin";

	Tensor3<float> core;
	core.read(filename);
	core_dim[0] = core.dim(0), core_dim[1] = core.dim(1), core_dim[2] = core.dim(2);
	int totalSize = core_dim[0] * core_dim[1] * core_dim[2];

	tmesh.resize(core_dim[2]);

	// unfold it
	Tensor2<float> tu0 = core.unfold(0), tu1 = core.unfold(1);

	PhGUtils::message("transferring the unfolded core tensor to GPU ...");

#if 1
	checkCudaErrors(hipHostAlloc((void**) &h_tu0, sizeof(float)*totalSize, hipHostMallocMapped));
	memcpy(h_tu0, tu0.rawptr(), sizeof(float)*totalSize);
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_tu0, h_tu0, 0));

	checkCudaErrors(hipHostAlloc((void**) &h_tu1, sizeof(float)*totalSize, hipHostMallocMapped));
	memcpy(h_tu1, tu1.rawptr(), sizeof(float)*totalSize);
	checkCudaErrors(hipHostGetDevicePointer((void**) &d_tu1, h_tu1, 0));
#else
	// transfer the unfolded core tensor to GPU
	checkCudaErrors(hipMalloc((void **) &d_tu0, sizeof(float)*totalSize));
	checkCudaErrors(hipMemcpy(d_tu0, tu0.rawptr(), sizeof(float)*totalSize, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void **) &d_tu1, sizeof(float)*totalSize));
	checkCudaErrors(hipMemcpy(d_tu1, tu1.rawptr(), sizeof(float)*totalSize, hipMemcpyHostToDevice));
#endif

	PhGUtils::message("done.");
	showCUDAMemoryUsage();

	PhGUtils::message("allocating memory for computation (tensors) ...");
	// allocate memory for the tm0, tm1, tm0RT, tm1RT, tplt
	checkCudaErrors(hipMalloc((void **) &d_tm0, sizeof(float)*core_dim[1]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tm0RT, sizeof(float)*core_dim[1]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tm1, sizeof(float)*core_dim[0]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tm1RT, sizeof(float)*core_dim[0]*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_tplt, sizeof(float)*core_dim[2]));
	checkCudaErrors(hipMalloc((void **) &d_mesh, sizeof(float)*core_dim[2]));
	checkCudaErrors(hipMemset(d_mesh, 0, sizeof(float)*core_dim[2]));
	showCUDAMemoryUsage();

	// read the prior
	PhGUtils::message("Loading prior data ...");

	// identity prior
	PhGUtils::message("Loading identity prior data ...");
	const string fnwid  = "../Data/blendshape/wid.bin";

	ifstream fwid(fnwid, ios::in | ios::binary );
	fwid.read(reinterpret_cast<char*>(&ndims_wid), sizeof(int));
	cout << "identity prior dim = " << ndims_wid << endl;
	vector<float> mu_wid0, mu_wid, sigma_wid;
	mu_wid0.resize(ndims_wid);
	mu_wid.resize(ndims_wid);
	sigma_wid.resize(ndims_wid*ndims_wid);

	fwid.read(reinterpret_cast<char*>(&(mu_wid0[0])), sizeof(float)*ndims_wid);
	fwid.read(reinterpret_cast<char*>(&(mu_wid[0])), sizeof(float)*ndims_wid);
	fwid.read(reinterpret_cast<char*>(&(sigma_wid[0])), sizeof(float)*ndims_wid*ndims_wid);

	fwid.close();

	PhGUtils::message("identity prior loaded.");
	PhGUtils::message("transferring identity prior to GPU ...");

	// transfer the identity prior to GPU
	checkCudaErrors(hipMalloc((void **) &d_mu_wid0, sizeof(float)*ndims_wid));
	checkCudaErrors(hipMemcpy(d_mu_wid0, &(mu_wid0[0]), sizeof(float)*ndims_wid, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void **) &d_mu_wid, sizeof(float)*ndims_wid));
	checkCudaErrors(hipMemcpy(d_mu_wid, &(mu_wid[0]), sizeof(float)*ndims_wid, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**) &d_mu_wid_weighted, sizeof(float)*ndims_wid));
	checkCudaErrors(hipMalloc((void**) &d_Wid, sizeof(float)*ndims_wid));

	checkCudaErrors(hipMalloc((void **) &d_sigma_wid, sizeof(float)*ndims_wid*ndims_wid));
	checkCudaErrors(hipMemcpy(d_sigma_wid, &(sigma_wid[0]), sizeof(float)*ndims_wid*ndims_wid, hipMemcpyHostToDevice));

	// write back for examiniation
	PhGUtils::write2file(sigma_wid, "wid.txt");

	PhGUtils::message("done.");
	showCUDAMemoryUsage();

	// expression prior
	PhGUtils::message("Loading expression prior data ...");
	const string fnwexp = "../Data/blendshape/wexp.bin";
	ifstream fwexp(fnwexp, ios::in | ios::binary );

	fwexp.read(reinterpret_cast<char*>(&ndims_wexp), sizeof(int));
	cout << "expression prior dim = " << ndims_wexp << endl;
	vector<float> mu_wexp0, mu_wexp, sigma_wexp;
	mu_wexp0.resize(ndims_wexp);
	mu_wexp.resize(ndims_wexp);
	sigma_wexp.resize(ndims_wexp*ndims_wexp);

	fwexp.read(reinterpret_cast<char*>(&(mu_wexp0[0])), sizeof(float)*ndims_wexp);
	fwexp.read(reinterpret_cast<char*>(&(mu_wexp[0])), sizeof(float)*ndims_wexp);
	fwexp.read(reinterpret_cast<char*>(&(sigma_wexp[0])), sizeof(float)*ndims_wexp*ndims_wexp);

	fwexp.close();

	PhGUtils::message("expression prior loaded.");
	PhGUtils::message("transferring expression prior to GPU ...");

	// transfer the expression prior to GPU
	checkCudaErrors(hipMalloc((void **) &d_mu_wexp0, sizeof(float)*ndims_wexp));
	checkCudaErrors(hipMemcpy(d_mu_wexp0, &(mu_wexp0[0]), sizeof(float)*ndims_wexp, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMalloc((void **) &d_mu_wexp, sizeof(float)*ndims_wexp));
	checkCudaErrors(hipMemcpy(d_mu_wexp, &(mu_wexp[0]), sizeof(float)*ndims_wexp, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**) &d_mu_wexp_weighted, sizeof(float)*ndims_wexp));
	checkCudaErrors(hipMalloc((void**) &d_Wexp, sizeof(float)*ndims_wexp));

	checkCudaErrors(hipMalloc((void **) &d_sigma_wexp, sizeof(float)*ndims_wexp*ndims_wexp));
	checkCudaErrors(hipMemcpy(d_sigma_wexp, &(sigma_wexp[0]), sizeof(float)*ndims_wexp*ndims_wexp, hipMemcpyHostToDevice));

	// write back for examination
	PhGUtils::write2file(sigma_wexp, "wexp.txt");

	PhGUtils::message("done.");
	showCUDAMemoryUsage();

	// load the indices of landmarks
	const string lmfn = "../Data/model/landmarks.txt";
	ifstream fin(lmfn, ios::in);
	if( fin.is_open() ) {
		landmarkIdx.reserve(128);
		int idx;
		while(fin.good()) {
			fin >> idx;
			landmarkIdx.push_back(idx);
		}
		PhGUtils::message("landmarks loaded.");
		cout << "total landmarks = " << landmarkIdx.size() << endl;
		ndims_fpts = landmarkIdx.size() * 3;
	}
	else {
		PhGUtils::abort("Failed to load landmarks!");
	}
	// allocate space for landmarks
	checkCudaErrors(hipMalloc((void**) &d_fptsIdx, sizeof(int)*landmarkIdx.size()));
	// upload the landmark indices
	checkCudaErrors(hipMemcpy(d_fptsIdx, &(landmarkIdx[0]), sizeof(int)*landmarkIdx.size(), hipMemcpyHostToDevice));

	h_q = new float[landmarkIdx.size()*3];
	checkCudaErrors(hipMalloc((void**) &d_q, sizeof(float)*landmarkIdx.size()*3));
	h_q2d = new float[landmarkIdx.size()*3];
	checkCudaErrors(hipMalloc((void**) &d_q2d, sizeof(float)*landmarkIdx.size()*3));

	ndims_pts = core_dim[2];	// constraints by the vertices, at most 3 constraints for each vertex

	checkCudaErrors(hipMalloc((void**) &d_targetLocations, sizeof(float)*ndims_pts));
	showCUDAMemoryUsage();

	PhGUtils::message("allocating memory for computataion ...");
	// allocate space for Aid, Aexp, AidtAid, AexptAexp, brhs, Aidtb, Aexptb
	checkCudaErrors(hipMalloc((void **) &d_RTparams, sizeof(float)*7));
	int maxParams = max(ndims_wid, ndims_wexp);
	checkCudaErrors(hipMalloc((void **) &d_A, sizeof(float)*(maxParams + ndims_fpts + ndims_pts) * maxParams));
	checkCudaErrors(hipMalloc((void **) &d_b, sizeof(float)*(ndims_pts + ndims_fpts + maxParams)));

	h_w_landmarks = new float[landmarkIdx.size()*3];
	checkCudaErrors(hipMalloc((void**) &d_w_landmarks, sizeof(float)*landmarkIdx.size()*3));

	checkCudaErrors(hipMalloc((void**) &d_icpc, sizeof(d_ICPConstraint)*MAX_ICPC_COUNT));
	checkCudaErrors(hipMalloc((void**) &d_nicpc, sizeof(int)));
	PhGUtils::message("done.");

	PhGUtils::message("allocating memory for incoming data ...");
	checkCudaErrors(hipMalloc((void**) &d_colordata, sizeof(unsigned char)*640*480*4));
	checkCudaErrors(hipMalloc((void**) &d_depthdata, sizeof(unsigned char)*640*480*4));

	checkCudaErrors(hipMalloc((void**) &d_indexMap, sizeof(unsigned char)*640*480*4));
	checkCudaErrors(hipMalloc((void**) &d_depthMap, sizeof(float)*640*480));
	PhGUtils::message("done.");

	showCUDAMemoryUsage();
}

__host__ void MultilinearReconstructorGPU::bindTarget(const vector<PhGUtils::Point3f>& pts)
{
	cout << "binding " << pts.size() << " targets ..." << endl;
	// update q array and q2d array on host side
	// they are stored in page-locked memory
	int npts = pts.size();
	for(int i=0;i<npts;i++) {
		int idx = i*3;
		h_q2d[idx] = pts[i].x, h_q2d[idx+1] = pts[i].y, h_q2d[idx+2] = pts[i].z;
		PhGUtils::colorToWorld(pts[i].x, pts[i].y, pts[i].z, h_q[idx], h_q[idx+1], h_q[idx+2]);
	}

	// compute depth mean and variance
	int validZcount = 0;
	float mu_depth = 0, sigma_depth = 0;
	for(int i=0;i<npts;i++) {
		float z = pts[i].z;
		if( z != 0 ){
			mu_depth += z;
			validZcount++;
		}
	}
	mu_depth /= validZcount;
	for(int i=0;i<npts;i++) {
		float z = pts[i].z;
		if( z != 0 ){
			float dz = z - mu_depth;
			sigma_depth += dz * dz;
		}
	}
	sigma_depth /= (validZcount-1);

	const float DEPTH_THRES = 1e-6;
	int validCount = 0;
	meanX = 0; meanY = 0; meanZ = 0;
	// initialize weights
	for(int i=0, idx=0;i<npts;i++, idx+=3) {
		const float3& p = make_float3(h_q[idx], h_q[idx+1], h_q[idx+2]);
		int isValid = (fabs(p.z) > DEPTH_THRES)?1:0;

		meanX += p.x * isValid;
		meanY += p.y * isValid;
		meanZ += p.z * isValid;

		float dz = p.z - mu_depth;
		float w_depth = exp(-fabs(dz) / (sigma_depth*50.0));

		// set the landmark weights
		h_w_landmarks[idx] = h_w_landmarks[idx+1] = h_w_landmarks[idx+2] = (i<64 || i>74)?isValid*w_depth:isValid*w_boundary*w_depth;
		validCount += isValid;
	}

	// upload to GPU
	PhGUtils::message("uploading targets to GPU ...");
	checkCudaErrors(hipMemcpy(d_q2d, h_q2d, sizeof(float)*npts*3, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_q, h_q, sizeof(float)*npts*3, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_w_landmarks, h_w_landmarks, sizeof(float)*npts*3, hipMemcpyHostToDevice));
	PhGUtils::message("done.");
}

__host__ void MultilinearReconstructorGPU::bindRGBDTarget(const vector<unsigned char>& colordata,
														  const vector<unsigned char>& depthdata) 
{
	PhGUtils::message("uploading image targets to GPU ...");

	// update both color data and depth data
	const int sz = sizeof(unsigned char)*640*480*4;
	checkCudaErrors(hipMemcpy(d_colordata, &(colordata[0]), sz, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_depthdata, &(depthdata[0]), sz, hipMemcpyHostToDevice));

	PhGUtils::message("done.");
}

__host__ void MultilinearReconstructorGPU::setBaseMesh(const PhGUtils::QuadMesh& m) {
	baseMesh = m;
	// upload the mesh topology
	int nfaces = baseMesh.faceCount();
	vector<int4> topo(nfaces);
	for(int i=0;i<nfaces;i++) {
		const PhGUtils::QuadMesh::face_t& f = baseMesh.face(i);
		topo[i] = make_int4(f.x, f.y, f.z, f.w);
	}

	PhGUtils::message("uploading mesh topology");
	cout << "face count = " << nfaces << endl;
	if( d_meshtopo ) {
		checkCudaErrors(hipFree(d_meshtopo));
	}
	checkCudaErrors(hipMalloc((void**) &d_meshtopo, sizeof(int4)*nfaces));
	checkCudaErrors(hipMemcpy(d_meshtopo, &(topo[0]), sizeof(int4)*nfaces, hipMemcpyHostToDevice));
	showCUDAMemoryUsage();
}

__host__ void MultilinearReconstructorGPU::initRenderer() {
	// off-screen rendering related
	depthMap.resize(640*480);
	indexMap.resize(640*480*4);
	mProj = PhGUtils::KinectColorProjection.transposed();
	mMv = PhGUtils::Matrix4x4f::identity();

	dummyWgt = shared_ptr<QGLWidget>(new QGLWidget());
	dummyWgt->hide();
	dummyWgt->makeCurrent();
	fbo = shared_ptr<QGLFramebufferObject>(new QGLFramebufferObject(640, 480, QGLFramebufferObject::Depth));
	dummyWgt->doneCurrent();
}

__host__ void MultilinearReconstructorGPU::fit(FittingOption op) {
	switch( op ) {
	case FIT_POSE:
		{
			fitPose();
			break;
		}
	case FIT_IDENTITY:
		{

			break;
		}
	case FIT_EXPRESSION:
		{

			break;
		}
	case FIT_POSE_AND_IDENTITY:
		{

			break;
		}
	case FIT_POSE_AND_EXPRESSION:
		{

			break;
		}
	case FIT_ALL:
		{

			break;
		}
	}
}

__host__ void MultilinearReconstructorGPU::fitPose() {
	cout << "fitting pose ..." << endl;
	
	// make rotation matrix and translation vector
	cout << "initial guess ..." << endl;
	PhGUtils::printArray(h_RTparams, 7);

	float errorThreshold_ICP = 1e-5;
	float errorDiffThreshold_ICP = errorThreshold * 1e-4;

	int iters = 0;
	float E0 = 0, E;
	bool converged = false;
	const int MaxIterations = 32;

	while( !converged && iters++<MaxIterations ) {
		transformMesh();
		updateMesh();
		renderMesh();
		int nicpc = collectICPConstraints(iters, MaxIterations);
		converged = fitRigidTransformation();
		E = computeError();
		PhGUtils::debug("iters", iters, "Error", E);

		// adaptive threshold
		converged |= E < (errorThreshold_ICP / (nicpc/5000.0));
		converged |= fabs(E - E0) < errorDiffThreshold_ICP;
		E0 = E;
	}

	// use the latest parameters
	transformMesh();
	updateMesh();
}

__host__ void MultilinearReconstructorGPU::fitPoseAndIdentity() {
}

__host__ void MultilinearReconstructorGPU::fitPoseAndExpression() {
}

__host__ void MultilinearReconstructorGPU::fitAll() {
}

__host__ void MultilinearReconstructorGPU::renderMesh()
{
	dummyWgt->makeCurrent();
	fbo->bind();

#if FBO_DEBUG_GPU
	cout << (fbo->isBound()?"bounded.":"not bounded.") << endl;
	cout << (fbo->isValid()?"valid.":"invalid.") << endl;
#endif

	glEnable(GL_DEPTH_TEST);
	glDepthMask(GL_TRUE);

	glEnable(GL_CULL_FACE);
	glCullFace(GL_BACK);

	glPushMatrix();

	// setup viewing parameters
	glViewport(0, 0, 640, 480);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glMultMatrixf(mProj.data());

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glMultMatrixf(mMv.data());

	glClearColor(0, 0, 0, 1);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT );

	glShadeModel(GL_SMOOTH);

	baseMesh.drawFaceIndices();	

	glReadPixels(0, 0, 640, 480, GL_DEPTH_COMPONENT, GL_FLOAT, &(depthMap[0]));
#if FBO_DEBUG_GPU
	GLenum errcode = glGetError();
	if (errcode != GL_NO_ERROR) {
		const GLubyte *errString = gluErrorString(errcode);
		fprintf (stderr, "OpenGL Error: %s\n", errString);
	}
#endif

	glReadPixels(0, 0, 640, 480, GL_RGBA, GL_UNSIGNED_BYTE, &(indexMap[0]));
#if FBO_DEBUG_GPU
	errcode = glGetError();
	if (errcode != GL_NO_ERROR) {
		const GLubyte *errString = gluErrorString(errcode);
		fprintf (stderr, "OpenGL Error: %s\n", errString);
	}
#endif

	glPopMatrix();

	glDisable(GL_CULL_FACE);

	fbo->release();
	dummyWgt->doneCurrent();

#if FBO_DEBUG_GPU
	ofstream fout("fbodepth.txt");
	PhGUtils::print2DArray(&(depthMap[0]), 480, 640, fout);
	fout.close();

	QImage img = PhGUtils::toQImage(&(indexMap[0]), 640, 480);	
	img.save("fbo.png");
#endif

	// upload result to GPU
	checkCudaErrors(hipMemcpy(d_indexMap, &indexMap[0], sizeof(unsigned char)*640*480*4, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_depthMap, &depthMap[0], sizeof(float)*640*480, hipMemcpyHostToDevice));
}

__global__ void clearICPConstraints(int* nicpc) {
	*nicpc = 0;
}

__device__ __forceinline__ float3 color2world(float u, float v, float d) {
	// focal length
	const float fx_rgb = 525.0, fy_rgb = 525.0;
	// for 640x480 image
	const float cx_rgb = 320.0, cy_rgb = 240.0;

	// This part is correct now.
	// Given a Kinect depth value, its depth in OpenGL coordinates
	// system must be negative.
	float depth = -d/1000.0;

	float3 res;
	// inverse mapping of projection
	res.x = -(u - cx_rgb) * depth / fx_rgb;
	res.y = (v - cy_rgb) * depth / fy_rgb;
	res.z = depth;
	return res;
}

__device__ __forceinline__ float3 world2color(float3 p) {
	// focal length
	const float fx_rgb = 525.0, fy_rgb = 525.0;
	// for 640x480 image
	const float cx_rgb = 320.0, cy_rgb = 240.0;

	float invZ = 1.0 / p.z;
	float3 uvd;
	uvd.x = clamp(cx_rgb - p.x * fx_rgb * invZ, 0.f, 639.f);
	uvd.y = clamp(cy_rgb + p.y * fy_rgb * invZ, 0.f, 479.f);
	uvd.z = -p.z*1000.0f;
	return uvd;
}

__device__ __forceinline__ int decodeIndex(unsigned char r, unsigned char g, unsigned char b) {
	int ri = r, gi = g, bi = b;
	return (ri << 16) | (gi << 8) | bi;
}

__device__ float point_to_triangle_distance(float3 p0, float3 p1, float3 p2, float3 p3, float3& hit) {
	float dist = 0;

	float3 d = p1 - p0;
	float3 e12 = p2 - p1, e13 = p3 - p1, e21 = -e12, e23 = p3 - p2, e31 = -e13, e32 = -e23;
	float3 e12n = normalize(e12), e13n = normalize(e13), e21n = -e12n, e23n = normalize(e23), e31n = -e13n, e32n = -e23n;

	float3 n = normalize(cross(e12, e13));

	float dDOTn = dot(d, n);
	float dnorm = length(d);
	float cosAlpha = dDOTn / dnorm;

	float dn = dDOTn * cosAlpha;
	float3 p0p0c = -dn * n;
	float3 p0c = p0 + p0p0c;

	float3 v1 = e21n + e31n, v2 = e12n + e32n, v3 = e13n + e23n;
	float3 p0cp1 = p1 - p0c, p0cp2 = p2 - p0c, p0cp3 = p3 - p0c;

	float3 c1 = cross(p0cp1, p0cp2), c2 = cross(p0cp2, p0cp3), c3 = cross(p0cp3, p0cp1);

	float d1 = dot(c1, n);
	float d2 = dot(c2, n);
	float d3 = dot(c3, n);

	float3 x0 = p0c, x1, x2;
	bool inside = true;
	if ( d1 < d2 && d1 < d3 && d1<0 )			//-- outside, p1, p2 side
	{
		inside = false;		
		x1 = p1; x2 = p2;
	}
	else if ( d2 < d1 && d2 < d3 && d2<0 )	//-- outside, p2, p3 side
	{
		inside = false;
		x1 = p2; x2 = p3;
	}
	else if ( d3 < d1 && d3 < d2 && d3<0 )	//-- outside, p3, p1 side
	{
		inside = false;
		x1 = p3; x2 = p1;
	}

	if (inside)
	{
		hit = p0c;
		dist = dn;
	}
	else
	{
		float3 x1x0 = x0 - x1, x2x0 = x0 - x1, x2x1 = x1 - x2;
		float L_x2x0 = length(x2x0);
		float t = dot(x1x0, x2x0) / (L_x2x0 * L_x2x0);

		hit = x1 + t * x2x1;

		float3 line = p0 - hit;
		dist = length(line);
	}

	return dist;

}

__device__ float3 compute_barycentric_coordinates(float3 p, float3 q1, float3 q2, float3 q3) {
	float3 e23 = q3 - q2, e21 = q1 - q2, e31 = q1 - q3;
	float3 d2 = p - q2, d3 = p - q3;
	float3 oriN = cross(e23, e21);
	float3 n = normalize(oriN);

	float invBTN = 1.0 / dot(oriN, n);
	float3 bcoord;
	bcoord.x = dot(cross(e23, d2), n) * invBTN;
	bcoord.y = dot(cross(e31, d3), n) * invBTN;
	bcoord.z = 1 - bcoord.x - bcoord.y;

	return bcoord;
}
//@note	need to upload the topology of the template mesh for constraint collection
__global__ void collectICPConstraints_kernel(
						float*				mesh,
						int4*				meshtopo,
						unsigned char*		indexMap,			// synthesized data
						float*				depthMap,			// synthesized data
						unsigned char*		colordata,			// capture data
						unsigned char*		depthdata,			// capture data
						d_ICPConstraint*	icpc,				// ICP constraints
						int*				nicpc,
						float thres
	) {
	float DIST_THRES = thres;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x > 639 || y > 479 ) return;

	int tid = y * 640 + x;

	int u = x, v = y;
	int idx = (v * 640 + u)*4;
	int vv = 479 - y;
	int didx = vv * 640 + u;
	
	if( depthMap[didx] < 1.0 ) {
		// valid pixel, see if it is a valid constraint
		float d = (depthdata[idx]<<16|depthdata[idx+1]<<8|depthdata[idx+2]);
		
		// bad pixel
		if( d == 0 ) return;

		// compute target location
		float3 q = color2world(u, v, d);

		// take a small window
		const int wSize = 2;
		int checkedFaces[(wSize+1)*(wSize+1)];
		int checkedCount = 0;
		float closestDist = FLT_MAX;
		int3 closestVerts;
		float3 closestHit;

		// check for the closest point face
		for(int r = max(v - wSize, 0); r <= min(v + wSize, 479); r++) {
			int rr = 479 - r;
			for(int c = max(u - wSize, 0); c <= min(u + wSize, 479); c++) {
				int pidx = rr * 640 + c;
				int poffset = pidx << 2;

				float depthVal = depthMap[pidx];
				if( depthVal < 1.0 ) {
					int fidx = decodeIndex(indexMap[poffset], indexMap[poffset+1], indexMap[poffset+2]);

					
					bool checked = false;
					// see if this face is already checked
					for(int j=0;j<checkedCount;j++) {
						if( fidx == checkedFaces[j] ){
							checked = true;
							break;
						}
					}
					if( checked ) continue;
					else {
						checkedFaces[checkedCount] = fidx;
						checkedCount++;
					}


					// not checked yet, check out this face
					int4 f = meshtopo[fidx];
					int4 vidx = f * 3;
					float3 v0 = make_float3(mesh[vidx.x], mesh[vidx.x+1], mesh[vidx.x+2]);
					float3 v1 = make_float3(mesh[vidx.y], mesh[vidx.y+1], mesh[vidx.y+2]);
					float3 v2 = make_float3(mesh[vidx.z], mesh[vidx.z+1], mesh[vidx.z+2]);
					float3 v3 = make_float3(mesh[vidx.w], mesh[vidx.w+1], mesh[vidx.w+2]);

					float3 hit1, hit2;
					float dist1 = point_to_triangle_distance(q, v0, v1, v2, hit1);
					float dist2 = point_to_triangle_distance(q, v1, v2, v3, hit2);
				
					// take the smaller one
					if( dist1 < dist2 && dist1 < closestDist) {
						closestDist = dist1;
						closestVerts.x = f.x, closestVerts.y = f.y, closestVerts.z = f.z;
						closestHit = hit1;
					}
					else if( dist2 < closestDist ) {
						closestDist = dist2;
						closestVerts.x = f.y, closestVerts.y = f.z, closestVerts.z = f.w;
						closestHit = hit2;
					}
				}
			}
		}

		if( closestDist < DIST_THRES ) {
			d_ICPConstraint cc;
			cc.q = q;
			cc.v = closestVerts;
			int3 vidx = cc.v*3;
			
			float3 v0 = make_float3(mesh[vidx.x], mesh[vidx.x+1], mesh[vidx.x+2]);
			float3 v1 = make_float3(mesh[vidx.y], mesh[vidx.y+1], mesh[vidx.y+2]);
			float3 v2 = make_float3(mesh[vidx.z], mesh[vidx.z+1], mesh[vidx.z+2]);
			
			cc.bcoords = compute_barycentric_coordinates( closestHit, v0, v1, v2 );
			int slot = atomicAdd(nicpc, 1);
			__threadfence();
			icpc[slot] = cc;
		}
	}
}

__host__ int MultilinearReconstructorGPU::collectICPConstraints(int iters, int maxIters) {
	const float DIST_THRES_MAX = 0.010;
	const float DIST_THRES_MIN = 0.001;
	float DIST_THRES = DIST_THRES_MAX + (DIST_THRES_MIN - DIST_THRES_MAX) * iters / (float)maxIters;
	PhGUtils::message("Collecting ICP constraints...");
	
	writeback(d_depthMap, 480, 640, "d_depthmap.txt");

	clearICPConstraints<<<1, 1, 0, mystream>>>(d_nicpc);
	checkCudaState();
	PhGUtils::Timer ticpc;
	ticpc.tic();
	dim3 block(16, 16, 1);
	dim3 grid(640/16, 480/16, 1);
	collectICPConstraints_kernel<<<grid, block, 0, mystream>>>( d_mesh,
																d_meshtopo,
																d_indexMap,
																d_depthMap,
																d_colordata,
																d_depthdata,
																d_icpc,
																d_nicpc,
																DIST_THRES);

	hipDeviceSynchronize();
	ticpc.toc("ICPC collection");
	checkCudaState();
	PhGUtils::message("ICPC computed.");
	// copy back the number of ICP constraints
	int icpcCount = 0;
	checkCudaErrors(hipMemcpy(&icpcCount, d_nicpc, sizeof(int), hipMemcpyDeviceToHost));
	cout << "ICPC = " << icpcCount << endl;
	
#if OUTPUT_ICPC
	vector<d_ICPConstraint> icpc(640*480);
	checkCudaErrors(hipMemcpy(&icpc[0], d_icpc, sizeof(d_ICPConstraint)*MAX_ICPC_COUNT, hipMemcpyDeviceToHost));
	ofstream fout("d_icpc.txt");
	for(int i=0;i<icpcCount;i++) {
		float3 bc = icpc[i].bcoords;
		int3 vidx = icpc[i].v * 3;
		float3 p;
		p.x = tmesh(vidx.x  ) * bc.x + tmesh(vidx.y  ) * bc.y + tmesh(vidx.z  ) * bc.z;
		p.y = tmesh(vidx.x+1) * bc.x + tmesh(vidx.y+1) * bc.y + tmesh(vidx.z+1) * bc.z;
		p.z = tmesh(vidx.x+2) * bc.x + tmesh(vidx.y+2) * bc.y + tmesh(vidx.z+2) * bc.z;
		fout << icpc[i].q.x << ' '
			 << icpc[i].q.y << ' '
			 << icpc[i].q.z << ' '
			 << p.x << ' '
			 << p.y << ' '
			 << p.z << ' '
			 << bc.x << ' '
			 << bc.y << ' '
			 << bc.z << endl;
	}
	fout.close();
#endif

	return icpcCount;
}

// use one dimensional configuration
__global__ void cost_ICP(float *unknowns, float *costfunc, int offset,
							 d_ICPConstraint* d_icpc, int nicpc,
							 float *d_tplt,
							 float w_ICP
							 ) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nicpc ) return;

	float s, rx, ry, rz, tx, ty, tz;
	rx = unknowns[0], ry = unknowns[1], rz = unknowns[2];
	tx = unknowns[3], ty = unknowns[4], tz = unknowns[5];
	s = unknowns[6];

	mat3 R = mat3::rotation(rx, ry, rz) * s;
	float3 T = make_float3(tx, ty, tz);

	d_ICPConstraint icpc = d_icpc[tid];
	const int3& v = icpc.v;
	const float3& bc = icpc.bcoords;

	int3 vidx = icpc.v*3;

	float3 v0 = make_float3(d_tplt[vidx.x], d_tplt[vidx.x+1], d_tplt[vidx.x+2]);
	float3 v1 = make_float3(d_tplt[vidx.y], d_tplt[vidx.y+1], d_tplt[vidx.y+2]);
	float3 v2 = make_float3(d_tplt[vidx.z], d_tplt[vidx.z+1], d_tplt[vidx.z+2]);

	const float3& q = icpc.q;

	float3 p = v0 * bc.x + v1 * bc.y + v2 * bc.z;
	p = R * p + T;

	costfunc[tid+offset] = length(p - q) * w_ICP;
}

__global__ void jacobian_ICP(float *unknowns, float *J, int offset,
							 d_ICPConstraint* d_icpc, int nicpc,
							 float *d_tplt,
							 float w_ICP) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nicpc ) return;

	float s, rx, ry, rz, tx, ty, tz;
	rx = unknowns[0], ry = unknowns[1], rz = unknowns[2];
	tx = unknowns[3], ty = unknowns[4], tz = unknowns[5];
	s = unknowns[6];

	mat3 R = mat3::rotation(rx, ry, rz) * s;
	float3 T = make_float3(tx, ty, tz);
	mat3 Jx, Jy, Jz;
	mat3::jacobian(rx, ry, rz, Jx, Jy, Jz);

	d_ICPConstraint icpc = d_icpc[tid];
	const int3& v = icpc.v;
	const float3& bc = icpc.bcoords;

	int3 vidx = icpc.v*3;

	float3 v0 = make_float3(d_tplt[vidx.x], d_tplt[vidx.x+1], d_tplt[vidx.x+2]);
	float3 v1 = make_float3(d_tplt[vidx.y], d_tplt[vidx.y+1], d_tplt[vidx.y+2]);
	float3 v2 = make_float3(d_tplt[vidx.z], d_tplt[vidx.z+1], d_tplt[vidx.z+2]);

	const float3& q = icpc.q;

	float3 p = v0 * bc.x + v1 * bc.y + v2 * bc.z;

	int jidx = tid*7+offset;

	// R * p
	float3 rp = R * p;

	// s * R * p + t - q
	float3 rk = s * rp + T - q;

	float3 jp = Jx * p;
	// \frac{\partial r_i}{\partial \theta_x}
	J[jidx++] = 2.0 * s * dot(jp, rk) * w_ICP;	

	jp = Jy * p;
	// \frac{\partial r_i}{\partial \theta_y}
	J[jidx++] = 2.0 * s * dot(jp, rk) * w_ICP;

	jp = Jz * p;
	// \frac{\partial r_i}{\partial \theta_z}
	J[jidx++] = 2.0 * s * dot(jp, rk) * w_ICP;

	// \frac{\partial r_i}{\partial \t_x}
	J[jidx++] = 2.0 * rk.x * w_ICP;

	// \frac{\partial r_i}{\partial \t_y}
	J[jidx++] = 2.0 * rk.y * w_ICP;

	// \frac{\partial r_i}{\partial \t_z}
	J[jidx++] = 2.0 * rk.z * w_ICP;

	// \frac{\partial r_i}{\partial s}
	J[jidx++] = 2.0 * dot(rp, rk) * w_ICP;
}

// use one dimensional configuration
/* @note	d_w_mask is		1.0				if i<42 || i > 74
							w_outer * w_fp	if 63 < i <= 74
							w_chin * w_fp	if 42 <= i <= 63
*/
__global__ void cost_FeaturePoints(float *unknowns, float *costfunc, int offset,
								   int *d_fptsIdx, float *d_q, float *d_q2d, int nfpts,
								   float *d_tplt,
								   float *d_w_landmarks, float *d_w_mask,
								   float w_fp_scale) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nfpts ) return;

	float s, rx, ry, rz, tx, ty, tz;
	rx = unknowns[0], ry = unknowns[1], rz = unknowns[2];
	tx = unknowns[3], ty = unknowns[4], tz = unknowns[5];
	s = unknowns[6];

	mat3 R = mat3::rotation(rx, ry, rz) * s;
	float3 T = make_float3(tx, ty, tz);

	int voffset = tid * 3;
	float wpt = d_w_landmarks[tid] * w_fp_scale * d_w_mask[tid];

	int vidx = d_fptsIdx[tid] * 3;
	float3 p = make_float3(d_tplt[vidx], d_tplt[vidx+1], d_tplt[vidx+2]);


	if( tid < 42 || tid > 74 ) {
		float3 q = make_float3(d_q[voffset], d_q[voffset+1], d_q[voffset+2]);
		costfunc[tid+offset] = length(p-q)*wpt;
	}
	else {
		float3 q = make_float3(d_q2d[voffset], d_q2d[voffset+1], d_q2d[voffset+2]);
		float3 uvd = world2color(p);
		float du = uvd.x - q.x, dv = uvd.y - q.y;
		costfunc[tid+offset] = (du*du+dv*dv)*wpt;
	}
}

__global__ void jacobian_FeaturePoints(float *unknowns, float *J, int offset,
								   int *d_fptsIdx, float *d_q, float *d_q2d, int nfpts,
								   float *d_tplt,
								   float *d_w_landmarks, float *d_w_mask,
								   float w_fp_scale) 
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= nfpts ) return;

	float s, rx, ry, rz, tx, ty, tz;
	rx = unknowns[0], ry = unknowns[1], rz = unknowns[2];
	tx = unknowns[3], ty = unknowns[4], tz = unknowns[5];
	s = unknowns[6];

	mat3 R = mat3::rotation(rx, ry, rz) * s;
	float3 T = make_float3(tx, ty, tz);
	mat3 Jx, Jy, Jz;
	mat3::jacobian(rx, ry, rz, Jx, Jy, Jz);

	int voffset = tid * 3;
	float wpt = d_w_landmarks[tid] * w_fp_scale * d_w_mask[tid];

	int vidx = d_fptsIdx[tid] * 3;
	int jidx = tid*7+offset;
	float3 p = make_float3(d_tplt[vidx], d_tplt[vidx+1], d_tplt[vidx+2]);

	if( tid < 42 || tid > 74 ) {
		float3 q = make_float3(d_q[voffset], d_q[voffset+1], d_q[voffset+2]);

		// R * p
		float3 rp = R * p;

		// s * R * p + t - q
		float3 rk = s * rp + T - q;

		float3 jp = Jx * p;
		// \frac{\partial r_i}{\partial \theta_x}
		J[jidx++] = 2.0 * s * dot(jp, rk) * wpt;	

		jp = Jy * p;
		// \frac{\partial r_i}{\partial \theta_y}
		J[jidx++] = 2.0 * s * dot(jp, rk) * wpt;

		jp = Jz * p;
		// \frac{\partial r_i}{\partial \theta_z}
		J[jidx++] = 2.0 * s * dot(jp, rk) * wpt;

		// \frac{\partial r_i}{\partial \t_x}
		J[jidx++] = 2.0 * rk.x * wpt;

		// \frac{\partial r_i}{\partial \t_y}
		J[jidx++] = 2.0 * rk.y * wpt;

		// \frac{\partial r_i}{\partial \t_z}
		J[jidx++] = 2.0 * rk.z * wpt;

		// \frac{\partial r_i}{\partial s}
		J[jidx++] = 2.0 * dot(rp, rk) * wpt;
	}
	else {
		float3 q = make_float3(d_q2d[voffset], d_q2d[voffset+1], d_q2d[voffset+2]);

		float3 rp = R * p;
		float3 pk = s * rp + T;

		float inv_z = 1.0 / pk.z;
		float inv_z2 = inv_z * inv_z;

		const float f_x = 525.0, f_y = 525.0;
		float Jf[6] = {0};
		Jf[0] = -f_x * inv_z; Jf[2] = f_x * pk.x * inv_z2;
		Jf[4] = f_y * inv_z; Jf[5] = -f_y * pk.y * inv_z2;

		float3 uvd = world2color(pk);
		float pu = uvd.x, pv = uvd.y, pd = uvd.z;

		// residue
		float rkx = pu - q.x, rky = pv - q.y;

		// J_? * p_k
		float3 jp = Jx * p;
		// J_f * J_? * p_k
		float jfjpx, jfjpy;
		
		jfjpx = Jf[0] * jp.x + Jf[2] * jp.z;
		jfjpy = Jf[4] * jp.y + Jf[5] * jp.z;
		// \frac{\partial r_i}{\partial \theta_x}
		J[jidx++] = 2.0 * s * (jfjpx * rkx + jfjpy * rky) * wpt;

		jp = Jy * p;
		jfjpx = Jf[0] * jp.x + Jf[2] * jp.z;
		jfjpy = Jf[4] * jp.y + Jf[5] * jp.z;
		// \frac{\partial r_i}{\partial \theta_y}
		J[jidx++] = 2.0 * s * (jfjpx * rkx + jfjpy * rky) * wpt;

		jp = Jz * p;
		jfjpx = Jf[0] * jp.x + Jf[2] * jp.z;
		jfjpy = Jf[4] * jp.y + Jf[5] * jp.z;
		// \frac{\partial r_i}{\partial \theta_z}
		J[jidx++] = 2.0 * s * (jfjpx * rkx + jfjpy * rky) * wpt;

		// \frac{\partial r_i}{\partial \t_x}
		J[jidx++] = 2.0 * (Jf[0] * rkx) * wpt;

		// \frac{\partial r_i}{\partial \t_y}
		J[jidx++] = 2.0 * (Jf[4] * rky) * wpt;

		// \frac{\partial r_i}{\partial \t_z}
		J[jidx++] = 2.0 * (Jf[2] * rkx + Jf[5] * rky) * wpt;

		// \frac{\partial r_i}{\partial s}
		jfjpx = Jf[0] * rp.x + Jf[2] * rp.z;
		jfjpy = Jf[4] * rp.y + Jf[5] * rp.z;
		J[jidx++] = 2.0 * (jfjpx * rkx + jfjpy * rky) * wpt;
	}
}

__global__ void cost_History() {
}

__global__ void jacobian_History() {
}

__host__ bool MultilinearReconstructorGPU::fitRigidTransformation() {
	// gauss-newton algorithm to estimate a new set of parameters
	int iters = GaussNewton();
	// update the parameters and check if convergence is obtained
	return true;
}

__global__ void computeError_kernel() {
}

__host__ float MultilinearReconstructorGPU::computeError() {
	return true;
}

// one dimensional configuration
// R is the rows of the rotation matrix
__device__ float3 R0, R1, R2, T;
__global__ void setupRigidTransformation(float r00, float r01, float r02,
										 float r10, float r11, float r12,
										 float r20, float r21, float r22,
										 float  t0, float  t1,  float t2)
{
	R0 = make_float3(r00, r01, r02);
	R1 = make_float3(r10, r11, r12);
	R2 = make_float3(r20, r21, r22);
	 T = make_float3( t0,  t1,  t2);

	printf("%f, %f, %f\n", R0.x, R0.y, R0.z);
	printf("%f, %f, %f\n", R1.x, R1.y, R1.z);
	printf("%f, %f, %f\n", R2.x, R2.y, R2.z);
}

__global__ void transformMesh_kernel(int nverts, float *d_tplt, float *d_mesh) 
{
	unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if( tid >= nverts ) return;

	unsigned int idx0 = tid*3;

	float3 p = make_float3(d_tplt[idx0], d_tplt[idx0+1], d_tplt[idx0+2]);

	d_mesh[idx0] = dot(R0, p) + T.x;
	d_mesh[idx0+1] = dot(R1, p) + T.y;
	d_mesh[idx0+2] = dot(R2, p) + T.z;
}

__host__ void MultilinearReconstructorGPU::transformMesh() {
	PhGUtils::Matrix3x3f Rot = PhGUtils::rotationMatrix(h_RTparams[0], h_RTparams[1], h_RTparams[2]) * h_RTparams[6];
	cout << Rot << endl;
	float3 Tvec = make_float3(h_RTparams[3], h_RTparams[4], h_RTparams[5]);
	cout << Tvec << endl;
	int npts = ndims_pts/3;
	
	checkCudaState();

	setupRigidTransformation<<<1, 1, 0, mystream>>>(Rot(0, 0), Rot(0, 1), Rot(0, 2), Rot(1, 0), Rot(1, 1), Rot(1, 2),
									   Rot(2, 0), Rot(2, 1), Rot(2, 2),	   Tvec.x,    Tvec.y,    Tvec.z);
	checkCudaState();

	cout << "npts = " << npts << endl;
	dim3 block(256, 1);
	dim3 grid((int)ceil(npts/(float)(block.x)), 1, 1);
	cout << "grid: " << grid.x << "x" << grid.y << endl;
	transformMesh_kernel<<<grid, block, 0, mystream>>>(npts, d_tplt, d_mesh);
	
	checkCudaState();
}

__host__ void MultilinearReconstructorGPU::updateMesh()
{
	cout << "mesh size = " << tmesh.length() << endl;
	cout << "device mesh address = " << d_mesh << endl;
	cout << "bytes to transfer = " << sizeof(float)*ndims_pts << endl;
	checkCudaErrors(hipMemcpy(tmesh.rawptr(), d_mesh, sizeof(float)*ndims_pts, hipMemcpyDeviceToHost));
	hipError_t err = hipDeviceSynchronize();
	checkCudaErrors(err);

	writeback(d_mesh, ndims_pts/3, 3, "d_mesh.txt");
	writeback(d_tplt, ndims_pts/3, 3, "d_tplt.txt");

	//#pragma omp parallel for
	for(int i=0;i<tmesh.length()/3;i++) {
		int idx = i * 3;
		baseMesh.vertex(i).x = tmesh(idx++);
		baseMesh.vertex(i).y = tmesh(idx++);
		baseMesh.vertex(i).z = tmesh(idx);
	}

#if 1
	PhGUtils::OBJWriter writer;
	writer.save(baseMesh, "../Data/tmesh.obj");
#endif
}